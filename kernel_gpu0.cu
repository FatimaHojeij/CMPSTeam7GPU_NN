#include "hip/hip_runtime.h"
#include <stdio.h>

#include "kernel.h"
#include "matrix.h"
#include "timer.h"


__global__ void spmspm(COOMatrix *result, unsigned int* nnz_out, CSRMatrix A){ 
	
	result->rowIdxs[0] = 1;
	result->colIdxs[0] = 1;
	result->values[0] = 5;
	*nnz_out = A->values[0];
}

void sparseNN(Vector* result, COOMatrix* outBuffer, COOMatrix** layerWeights, float bias, unsigned int numLayers) {

	CSCMatrix* W[numLayers];
	for (unsigned int layer = 0; layer < numLayers; ++layer) {
			W[layer] = createCSCfromCOO(layerWeights[layer]);
	}
	stopTimeAndPrint(&timer, "Convert weights to CSC");
    //outBuffer_d allocation
	COOMatrix *outBuffer_d; 
	unsigned int* out_rowIdxs_d;
	unsigned int* out_colIdxs_d;
	float* out_values_d;
	unsigned int* out_nnz_d;
	unsigned int* out_nnz_h = (unsigned int*) malloc(sizeof(unsigned int*));
	*out_nnz_h = outBuffer->nnz;
	hipMalloc((void**)&outBuffer_d, sizeof(COOMatrix));
	hipMalloc((void**)&out_rowIdxs_d, outBuffer->capacity * sizeof(unsigned int));
	hipMalloc((void**)&out_colIdxs_d, outBuffer->capacity * sizeof(unsigned int));
	hipMalloc((void**)&out_values_d, outBuffer->capacity * sizeof(float));
	hipMalloc((void**)&out_nnz_d, sizeof(unsigned int));



	//copying outbuffer
	hipMemcpy(outBuffer_d, outBuffer, sizeof(COOMatrix), hipMemcpyHostToDevice);
	hipMemcpy(out_rowIdxs_d, outBuffer->rowIdxs, outBuffer->capacity * sizeof(unsigned int), hipMemcpyHostToDevice);
	hipMemcpy(out_colIdxs_d, outBuffer->colIdxs, outBuffer->capacity * sizeof(unsigned int), hipMemcpyHostToDevice);
	hipMemcpy(out_values_d, outBuffer->values, outBuffer->capacity * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(out_nnz_d, out_nnz_h, sizeof(unsigned int), hipMemcpyHostToDevice);
	hipMemcpy(out_nnz_d, &(outBuffer->nnz), sizeof(unsigned int), hipMemcpyHostToDevice);
	hipMemcpy(&(outBuffer_d->rowIdxs), &out_rowIdxs_d, sizeof(unsigned int*), hipMemcpyHostToDevice);
	hipMemcpy(&(outBuffer_d->colIdxs), &out_colIdxs_d, sizeof(unsigned int*), hipMemcpyHostToDevice);
	hipMemcpy(&(outBuffer_d->values), &out_values_d, sizeof(float*), hipMemcpyHostToDevice);
	hipMemcpy(&out_nnz_d, &out_nnz_h, sizeof(unsigned int*), hipMemcpyHostToDevice);
	hipDeviceSynchronize();
	printf("nnz before kernel call %d \n", outBuffer->nnz);

	unsigned int layer = 0;
	CSCMatrix* W_d;
		unsigned int* w_colPtrs_d;
		unsigned int* w_rowIdxs_d;
		float* w_values_d;
		hipMalloc((void**)&W_d, sizeof(CSCMatrix));
        hipMalloc((void**)&w_colPtrs_d, (W[layer]->numCols + 1)* sizeof(unsigned int));
        hipMalloc((void**)&w_rowIdxs_d, W[layer]->numRows * sizeof(unsigned int));
        hipMalloc((void**)&w_values_d, W[layer]->numRows * sizeof(float));
		//copying W_d[layer]
		hipMemcpy(W_d, W[layer], sizeof(CSCMatrix), hipMemcpyHostToDevice);
		hipMemcpy(w_colPtrs_d, W[layer]->colPtrs, W[layer]->numCols * sizeof(unsigned int), hipMemcpyHostToDevice);
        hipMemcpy(w_rowIdxs_d, W[layer]->rowIdxs, W[layer]->numRows * sizeof(unsigned int), hipMemcpyHostToDevice);
        hipMemcpy(w_values_d, W[layer]->values, W[layer]->numRows * sizeof(float), hipMemcpyHostToDevice);
		hipMemcpy(&(W_d->colPtrs), &w_colPtrs_d, sizeof(unsigned int*), hipMemcpyHostToDevice);
		hipMemcpy(&(W_d->rowIdxs), &w_rowIdxs_d, sizeof(unsigned int*), hipMemcpyHostToDevice);
		hipMemcpy(&(W_d->values), &w_values_d, sizeof(float*), hipMemcpyHostToDevice);
		
	spmspm <<<1, 1>>> (outBuffer_d, out_nnz_d, *W_d);
	hipDeviceSynchronize();

	//copy back       
	hipMemcpy(outBuffer->rowIdxs, out_rowIdxs_d, outBuffer->capacity * sizeof(unsigned int), hipMemcpyDeviceToHost);
	hipMemcpy(outBuffer->colIdxs, out_colIdxs_d, outBuffer->capacity * sizeof(unsigned int), hipMemcpyDeviceToHost);
	hipMemcpy(outBuffer->values, out_values_d, outBuffer->capacity * sizeof(float), hipMemcpyDeviceToHost);
	hipMemcpy(out_nnz_h, out_nnz_d, sizeof(unsigned int), hipMemcpyDeviceToHost);
	hipDeviceSynchronize();
	outBuffer->nnz = *out_nnz_h;
	printf("%f \n", outBuffer->values[0]);
	printf("nnz after kernel call %d \n", outBuffer->nnz);



}
