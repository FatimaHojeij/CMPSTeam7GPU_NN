#include "hip/hip_runtime.h"

#include <stdio.h>

#include "kernel.h"
#include "matrix.h"
#include "timer.h"


#define THRESHOLD 0.000001
#define YMAX 32
#define threads 1024


__global__ void spmspm(CSRMatrix *result, CSRMatrix *A, CSCMatrix *B, float bias) {


	

}

//converts from CSRMatrix to Vector and a vector of indices where the row is not all zeros
void findNonzeroRows(Vector* v, CSRMatrix* A) {
	unsigned int nnz = 0;
	for (unsigned int r = 0; r < A->numRows; ++r) {
		unsigned int rowPtrA = A->rowPtrs[r];
		unsigned int nnzA = A->rowPtrs[r + 1] - rowPtrA;
		if (nnzA > 0) {
			if (nnz >= v->capacity) {
				expandVectorCapacity(v, 2 * v->capacity);
			}
			v->data[nnz] = r;
			++nnz;
		}
	}
	v->nnz = nnz;
}

void sparseNN(Vector* result, COOMatrix* featureVectors, COOMatrix** layerWeights, float bias, unsigned int numLayers) {
	
	Timer timer;
	
	// Convert featureVectors to CSR
	startTime(&timer);
	CSRMatrix* Y0 = createCSRfromCOO(featureVectors);
	stopTimeAndPrint(&timer, "Convert feature vectors to CSR");
	
	// Convert layer weights to CSC
	startTime(&timer);
	CSCMatrix* W[numLayers];
	for (unsigned int layer = 0; layer < numLayers; ++layer) {
		W[layer] = createCSCfromCOO(layerWeights[layer]);
	}
	stopTimeAndPrint(&timer, "Convert weights to CSC");

	// Double buffers
	startTime(&timer);
	CSRMatrix *tmp = createEmptyCSR(Y0->numRows, Y0->numCols, 2 * Y0->nnz);
	CSRMatrix *inBuffer = Y0;
	CSRMatrix *outBuffer = tmp;
	stopTimeAndPrint(&timer, "Allocate temporary buffer");



	// Allocate GPU memory
	startTime(&timer);

	//result_d allocation
	//Vector *result_d;
	//result_d.nnz = result.nnz;
	//result_d.capacity = result.capacity;
	//hipMalloc((void**)&result_d.data, result.capacity * sizeof(unsigned int));

	//inBuffer_d allocation
	CSRMatrix *inBuffer_d;
	inBuffer_d.numRows = inBuffer.numRows;
	inBuffer_d.numCols = inBuffer.numCols;
	inBuffer_d.nnz = inBuffer.nnz;
	inBuffer_d.capacity = inBuffer.capacity;
	hipMalloc((void**)&inBuffer_d.rowPtrs, (inBuffer_d.numRows + 1) * sizeof(unsigned int));
	hipMalloc((void**)&inBuffer_d.colIdxs, inBuffer_d.numCols * sizeof(unsigned int));
	hipMalloc((void**)&inBuffer_d.values, inBuffer_d.numCols * sizeof(float));

	//outBuffer_d allocation
	CSRMatrix *outBuffer_d;
	outBuffer_d.numRows = outBuffer.numRows;
	outBuffer_d.numCols = outBuffer.numCols;
	outBuffer_d.nnz = outBuffer.nnz;
	outBuffer_d.capacity = outBuffer.capacity;
	hipMalloc((void**)&outBuffer_d.rowPtrs, (outBuffer_d.numRows + 1) * sizeof(unsigned int));
	hipMalloc((void**)&outBuffer_d.colIdxs, outBuffer_d.numCols * sizeof(unsigned int));
	hipMalloc((void**)&outBuffer_d.values, outBuffer_d.numCols * sizeof(float));

	// allocating W_d 
	CSCMatrix* W_d[numLayers];
	for (unsigned int layer = 0; layer < numLayers; ++layer) {
		W_d[layer].numRows = W[layer].numRows;
		W_d[layer].numCols = W[layer].numCols;
		W_d[layer].nnz = W[layer].nnz;
		W_d[layer].capacity = W[layer].capacity;
		hipMalloc((void**)&W[layer]_d.colPtrs, W[layer]_d.numCols * sizeof(unsigned int));
		hipMalloc((void**)&W[layer]_d.rowIdxs, W[layer_d].numRows * sizeof(unsigned int));
		hipMalloc((void**)&W[layer]_d.values, W[layer_d].numRows * sizeof(float));
	}

	hipDeviceSynchronize();
	stopTime(&timer);
	printElapsedTime(timer, "Allocation time on GPU Memory");

	// Copy data to GPU
	startTime(&timer);
	
	//for result
	hipMemcpy(result_d.data, result.data, result_d.capacity * sizeof(unsigned int), hipMemcpyHostToDevice);
	

	//for inbuffer
	hipMemcpy(inBuffer_d.rowPtrs, inBuffer.rowPtrs, inBuffer_d.numRows * sizeof(unsigned int), hipMemcpyHostToDevice);
	hipMemcpy(inBuffer_d.colIdxs, inBuffer.colIdxs, inBuffer_d.numCols * sizeof(unsigned int), hipMemcpyHostToDevice);
	hipMemcpy(inBuffer_d.values, inBuffer.values, inBuffer_d.numCols * sizeof(float), hipMemcpyHostToDevice);

	//for outbuffer
	hipMemcpy(outBuffer_d.rowPtrs, outBuffer.rowPtrs, outBuffer_d.numRows * sizeof(unsigned int), hipMemcpyHostToDevice);
	hipMemcpy(outBuffer_d.colIdxs, outBuffer.colIdxs, outBuffer_d.numCols * sizeof(unsigned int), hipMemcpyHostToDevice);
	hipMemcpy(outBuffer_d.values, outBuffer.values, outBuffer_d.numCols * sizeof(float), hipMemcpyHostToDevice);

	//for Weights
	for (unsigned int layer = 0; layer < numLayers; ++layer) {
		hipMemcpy(W_d[layer].colPtrs, W[layer].colPtrs, W_d[layer].numCols * sizeof(unsigned int), hipMemcpyHostToDevice);
		hipMemcpy(W_d[layer].rowIdxs, W[layer].rowIdxs, W_d[layer].numRows * sizeof(unsigned int), hipMemcpyHostToDevice);
		hipMemcpy(W_d[layer].values, W[layer].values, W_d[layer].numRows * sizeof(float), hipMemcpyHostToDevice);
	}

	hipDeviceSynchronize();
	stopTime(&timer);
	printElapsedTime(timer, "Copy to GPU time");

	//kernel loop

	// Loop over layers
	for (unsigned int layer = 0; layer < numLayers; ++layer) {

		// SpMSpM
		printf("Computing layer %u (SpMSpM)", layer);
		startTime(&timer);
		
		
		//do kernel call instead
		int outputSize = inBuffer_d->numRows * W_d[layer]->numCols;
		int numThreadsPerBlock = threads;
		int numBlocks = (outputSize + numThreadsPerBlock - 1)/numThreadsPerBlock ;
		spmspm <<<numBlocks, numThreadsPerBlock>>> (outBuffer_d,inBuffer_d,W_d[layer],bias);
		
		hipDeviceSynchronize();
		stopTimeAndPrint(&timer, "");

		// Swap buffers
		CSRMatrix *t = inBuffer_d;
		inBuffer_d = outBuffer_d;
		outBuffer_d = t;

	}

	
	// Copy data from GPU
	startTime(&timer);

	// TODO
	
	hipMemcpy(inBuffer.rowPtrs, inBuffer_d.rowPtrs, inBuffer_d.numRows * sizeof(unsigned int), hipMemcpyDeviceToHost);
	hipMemcpy(inBuffer.colIdxs, inBuffer_d.colIdxs, inBuffer_d.numCols * sizeof(unsigned int), hipMemcpyDeviceToHost);
	hipMemcpy(inBuffer.values, inBuffer_d.values, inBuffer_d.numCols * sizeof(float), hipMemcpyDeviceToHost);


	hipDeviceSynchronize();
	stopTime(&timer);
	printElapsedTime(timer, "Copy from GPU time");

	//CPU 
	// Find nonzero rows
	startTime(&timer);
	findNonzeroRows(result, inBuffer);
	stopTimeAndPrint(&timer, "Find nonzero rows");

	// Free GPU memory
	startTime(&timer);

	hipFree(inBuffer_d.rowPtrs);
	hipFree(inBuffer_d.colIdxs);
	hipFree(inBuffer_d.values);
	hipFree(outBuffer_d.rowPtrs);
	hipFree(outBuffer_d.colIdxs);
	hipFree(outBuffer_d.values);
	for (unsigned int layer = 0; layer < numLayers; ++layer) {
		hipFree(W_d[layer].colPtrs);
		hipFree(W_d[layer].rowIdxs);
		hipFree(W_d[layer].values);

	}


	hipDeviceSynchronize();
	stopTime(&timer);

	printElapsedTime(timer, "Deallocation time");

	// Free buffers
	startTime(&timer);
	freeCSR(Y0);
	for (unsigned int layer = 0; layer < numLayers; ++layer) {
		freeCSC(W[layer]);
	}
	freeCSR(tmp);
	stopTimeAndPrint(&timer, "Deallocate memory");
	

}

