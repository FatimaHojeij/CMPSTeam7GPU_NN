#include "hip/hip_runtime.h"
#include <stdio.h>

#include "kernel.h"
#include "matrix.h"
#include "timer.h"


#define THRESHOLD 0.000001
#define YMAX 32
#define threads 512


__global__ void spmspm(COOMatrix *result, CSRMatrix A, CSCMatrix B, float bias) {
    unsigned int r= blockIdx.y*blockDim.y +threadIdx.y;
    unsigned int c= blockIdx.x*blockDim.x + threadIdx.x;
    unsigned int rowPtrA;
    unsigned int nnzA;
        if(r < A.numRows && c < B.numCols){
                rowPtrA = A.rowPtrs[r];
                nnzA = A.rowPtrs[r + 1] - rowPtrA;
                if(nnzA>0) { // if a row is not all zeros , we do computation otherwise we skip row
                        //ptrs to cols and vals of A[r]
                        unsigned int* colIdxsA = A.colIdxs + rowPtrA;
                        float* valueA = A.values + rowPtrA;
                        //we will take one column of B
                        unsigned int colPtrB = B.colPtrs[c];
                        unsigned int nnzB = B.colPtrs[c + 1] - colPtrB;
                        if(nnzB>0) { // if a col in B is not all zeros, we do computation otherwise skip//ptrs to rows and vals of B[c]
                                unsigned int* rowIdxsB = B.rowIdxs + colPtrB;
                                float* valueB = B.values + colPtrB;
                                // Loop and find intersection
                                float sum = 0.0f;
                                unsigned int ia = 0, ib = 0;
                                while(ia < nnzA && ib < nnzB) { // loops over all non zeros from A and B and stop when there is no more non zero
                                        unsigned int colIdx = colIdxsA[ia]; //single item col index from A
                                        unsigned int rowIdx = rowIdxsB[ib]; //single item row index from B
                                        if(colIdx < rowIdx) {
                                                ia++;
                                        } else if(colIdx > rowIdx) {
                                                ib++;
                                        } else {
                                                sum += valueA[ia]*valueB[ib];// do the multiplication of the row that matches the column
                                                ia++;
                                                ib++;
                                        }
                                }
                                if(sum > THRESHOLD || sum < -THRESHOLD) { //if not smaller than abs(threshold)
                                        sum += bias; //add to it the bias
                                        //Remove negative and zero values
                                        if(sum > 0) {//if end result is positive otherwise I also do not want to add it to result
                                                if(sum>YMAX) { //make sure it is on an upper limit
                                                        sum = YMAX;
                                                }
                                                unsigned int nnzIndxTemp = atomicAdd(&(result->nnz),1); //counts how many non zero elements I have
                                                result->rowIdxs[nnzIndxTemp] = r;
                                                result->colIdxs[nnzIndxTemp] = c;
                                                result->values[nnzIndxTemp] = sum;
                                        }
                                }
                        }

                }
        }
}

COOMatrix* sortCOO(COOMatrix *A){

        // sorting rows
         for (unsigned int i = 0; i < A->nnz; i++)
                for (unsigned int j = 0; j < A->nnz-i-1; j++)
                {    if (A->rowIdxs[j] > A->rowIdxs[j+1]){
                                unsigned int r = A->rowIdxs[j];
                                unsigned int c =  A->colIdxs[j];
                                float v = A->values[j];
                                A->rowIdxs[j] = A->rowIdxs[j+1];
                                A->colIdxs[j] = A->colIdxs[j+1];
                                A->values[j] = A->values[j+1];
                                A->rowIdxs[j+1] = r;
                                A->colIdxs[j+1] = c;
                                A->values[j+1] = v;
                        }
                }

         // sorting the col
        // int count = 0;
         int begin = 0;
         for(unsigned int i  = 0 ;  i < A->nnz -1 ; i++)
         {
                 //count++;
                 if(A->rowIdxs[i] != A->rowIdxs[i+1])
                 {
                         //sort the col
                        for(int k = begin ;  k< i + begin; k++)
                                for (int m = begin ; m < i + begin - k -1 ;m++)
                                        if(A->colIdxs[m] > A->colIdxs[m+1]){
                                                unsigned int c = A->colIdxs[m];
                                                float v = A->values[m];
                                                A->colIdxs[m] = A->colIdxs[m+1];
                                                A->values[m] = A->values[m+1];
                                                A->colIdxs[m+1] =c;
                                                A->values[m+1] = v;

                                        }

                        // count = 0;
                        begin= i+1;
                }


        }
        return A;



 }
 
 //converts from CSRMatrix to Vector and a vector of indices where the row is not all zeros
void findNonzeroRows(Vector* v, CSRMatrix* A) {
        unsigned int nnz = 0;
        for (unsigned int r = 0; r < A->numRows; ++r) {
                unsigned int rowPtrA = A->rowPtrs[r];
                unsigned int nnzA = A->rowPtrs[r + 1] - rowPtrA;
                if (nnzA > 0) {
                        if (nnz >= v->capacity) {
                                expandVectorCapacity(v, 2 * v->capacity);
                        }
                        v->data[nnz] = r;
                        ++nnz;
                }
        }
        v->nnz = nnz;
}

COOMatrix* createEmptyCOO(unsigned int numRows, unsigned int numCols, unsigned int capacity) {
        COOMatrix *coo = (COOMatrix *)malloc(sizeof(COOMatrix));
        coo->rowIdxs= (unsigned int *)malloc(capacity * sizeof(unsigned int));
        coo->colIdxs= (unsigned int *)malloc(capacity * sizeof(unsigned int));
        coo->values= (float *)malloc( capacity * sizeof(float));
        coo->numRows = numRows;
        coo->numCols = numCols;
        coo->nnz = 0;
        coo->capacity = capacity;
        return coo;
}

void sparseNN(Vector* result, COOMatrix* featureVectors, COOMatrix** layerWeights, float bias, unsigned int numLayers) {
	Timer timer;

    // Convert featureVectors to CSR
    startTime(&timer);
	CSRMatrix* Y0 = createCSRfromCOO(featureVectors);
	stopTimeAndPrint(&timer, "Convert feature vectors to CSR");

	// Convert layer weights to CSC
	startTime(&timer);
	CSCMatrix* W[numLayers];
	for (unsigned int layer = 0; layer < numLayers; ++layer) {
			W[layer] = createCSCfromCOO(layerWeights[layer]);
	}
	stopTimeAndPrint(&timer, "Convert weights to CSC");

	// Double buffers
	startTime(&timer);
	COOMatrix *tmp = createEmptyCOO(Y0->numRows, Y0->numCols, 5 * Y0->nnz);
	CSRMatrix *inBuffer = Y0;
	COOMatrix *outBuffer = tmp;
	stopTimeAndPrint(&timer, "Allocate temporary buffer");



    // Allocate GPU memory
    startTime(&timer);
	
	
	//inBuffer_d allocation
	CSRMatrix* inBuffer_d;
    unsigned int* in_rowPtrs_d;
    unsigned int* in_colIdxs_d;
    float* in_values_d;
	hipMalloc((void**) &inBuffer_d, sizeof(CSRMatrix));
    hipMalloc((void**) &in_rowPtrs_d, (inBuffer->numRows + 1) * sizeof(unsigned int));
    hipMalloc((void**) &in_colIdxs_d, inBuffer->numCols * sizeof(unsigned int));
    hipMalloc((void**) &in_values_d, inBuffer->numCols * sizeof(float));
	
	
	//outBuffer_d allocation
    COOMatrix *outBuffer_d;
	unsigned int* out_rowIdxs_d;
    unsigned int* out_colIdxs_d;
    float* out_values_d;
    hipMalloc((void**)&outBuffer_d, sizeof(COOMatrix));
    hipMalloc((void**)&out_rowIdxs_d, outBuffer->capacity * sizeof(unsigned int));
    hipMalloc((void**)&out_colIdxs_d, outBuffer->capacity * sizeof(unsigned int));
    hipMalloc((void**)&out_values_d, outBuffer->capacity * sizeof(float));
		
	
	
	//copying inbuffer
	hipMemcpy(inBuffer_d, inBuffer, sizeof(CSRMatrix), hipMemcpyHostToDevice);
	hipMemcpy(in_rowPtrs_d, inBuffer->rowPtrs, (inBuffer->numRows + 1) * sizeof(unsigned int), hipMemcpyHostToDevice);
	hipMemcpy(in_colIdxs_d, inBuffer->colIdxs, inBuffer->numCols * sizeof(unsigned int), hipMemcpyHostToDevice);
	hipMemcpy(in_values_d, inBuffer->values, inBuffer->numCols * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(&(inBuffer_d->rowPtrs), &in_rowPtrs_d, sizeof(unsigned int*), hipMemcpyHostToDevice);
	hipMemcpy(&(inBuffer_d->colIdxs), &in_colIdxs_d, sizeof(unsigned int*), hipMemcpyHostToDevice);
	hipMemcpy(&(inBuffer_d->values), &in_values_d, sizeof(float*), hipMemcpyHostToDevice);
	printElapsedTime(timer, "For inBuffer");
	
	//copying outbuffer
    hipMemcpy(outBuffer_d, outBuffer, sizeof(COOMatrix), hipMemcpyHostToDevice);
	hipMemcpy(out_rowIdxs_d, outBuffer->rowIdxs, outBuffer->capacity * sizeof(unsigned int), hipMemcpyHostToDevice);
    hipMemcpy(out_colIdxs_d, outBuffer->colIdxs, outBuffer->capacity * sizeof(unsigned int), hipMemcpyHostToDevice);
    hipMemcpy(out_values_d, outBuffer->values, outBuffer->capacity * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(&(outBuffer_d->rowIdxs), &out_rowIdxs_d, sizeof(unsigned int*), hipMemcpyHostToDevice);
	hipMemcpy(&(outBuffer_d->colIdxs), &out_colIdxs_d, sizeof(unsigned int*), hipMemcpyHostToDevice);
	hipMemcpy(&(outBuffer_d->values), &out_values_d, sizeof(float*), hipMemcpyHostToDevice);
    printElapsedTime(timer, "For outBuffer");
	
	hipDeviceSynchronize();
    stopTime(&timer);
    printElapsedTime(timer, "Allocation & Copy to GPU time");
	
	
	for (unsigned int layer = 0; layer < numLayers; ++layer) {
		CSCMatrix* W_d;
		unsigned int* w_colPtrs_d;
		unsigned int* w_rowIdxs_d;
		float* w_values_d;
		hipMalloc((void**)&W_d, sizeof(CSCMatrix));
        hipMalloc((void**)&w_colPtrs_d, (W[layer]->numCols + 1)* sizeof(unsigned int));
        hipMalloc((void**)&w_rowIdxs_d, W[layer]->numRows * sizeof(unsigned int));
        hipMalloc((void**)&w_values_d, W[layer]->numRows * sizeof(float));
		//copying W_d[layer]
		hipMemcpy(W_d, W[layer], sizeof(CSCMatrix), hipMemcpyHostToDevice);
		hipMemcpy(w_colPtrs_d, W[layer]->colPtrs, W[layer]->numCols * sizeof(unsigned int), hipMemcpyHostToDevice);
        hipMemcpy(w_rowIdxs_d, W[layer]->rowIdxs, W[layer]->numRows * sizeof(unsigned int), hipMemcpyHostToDevice);
        hipMemcpy(w_values_d, W[layer]->values, W[layer]->numRows * sizeof(float), hipMemcpyHostToDevice);
		hipMemcpy(&(W_d->colPtrs), &w_colPtrs_d, sizeof(unsigned int*), hipMemcpyHostToDevice);
		hipMemcpy(&(W_d->rowIdxs), &w_rowIdxs_d, sizeof(unsigned int*), hipMemcpyHostToDevice);
		hipMemcpy(&(W_d->values), &w_values_d, sizeof(float*), hipMemcpyHostToDevice);
		
		
		dim3 numThreadsPerBlock(threads, threads);
        dim3 numBlocks((W[layer]->numCols + numThreadsPerBlock.x - 1)/numThreadsPerBlock.x,(inBuffer_d.numRows + numThreadsPerBlock.y - 1)/numThreadsPerBlock.y);
        spmspm <<<numBlocks, numThreadsPerBlock>>> (outBuffer_d, *inBuffer_d, *W_d, bias);
        hipDeviceSynchronize();
        stopTimeAndPrint(&timer, "");
		
		stopTimeAndPrint(&timer, "For Out Buffer");
		hipMemcpy(outBuffer, outBuffer_d, sizeof(COOMatrix), hipMemcpyDeviceToHost);
		//struct fields as variables(?)
		hipMemcpy(outBuffer->rowIdxs, out_rowIdxs_d, outBuffer->capacity * sizeof(unsigned int), hipMemcpyDeviceToHost);
		hipMemcpy(outBuffer->colIdxs, out_colIdxs_d, outBuffer->capacity * sizeof(unsigned int), hipMemcpyDeviceToHost);
		hipMemcpy(outBuffer->values, out_values_d, outBuffer->capacity * sizeof(float), hipMemcpyDeviceToHost);
		
		
		
		stopTimeAndPrint(&timer, "For Sort");
       		inBuffer = createCSRfromCOO(sortCOO(outBuffer));
       		stopTimeAndPrint(&timer, "Out of sort");
		
		//do we need to malloc again (?)
		hipMemcpy(inBuffer_d, inBuffer, sizeof(CSRMatrix), hipMemcpyHostToDevice);
		hipMemcpy(in_rowPtrs_d, inBuffer->rowPtrs, (inBuffer->numRows + 1) * sizeof(unsigned int), hipMemcpyHostToDevice);
		hipMemcpy(in_colIdxs_d, inBuffer->colIdxs, inBuffer->numCols * sizeof(unsigned int), hipMemcpyHostToDevice);
		hipMemcpy(in_values_d, inBuffer->values, inBuffer->numCols * sizeof(float), hipMemcpyHostToDevice);
		hipMemcpy(&(inBuffer_d->rowPtrs), &in_rowPtrs_d, sizeof(unsigned int*), hipMemcpyHostToDevice);
		hipMemcpy(&(inBuffer_d->colIdxs), &in_colIdxs_d, sizeof(unsigned int*), hipMemcpyHostToDevice);
		hipMemcpy(&(inBuffer_d->values), &in_values_d, sizeof(float*), hipMemcpyHostToDevice);
		
		
		outBuffer = createEmptyCOO(inBuffer->numRows, inBuffer->numCols, 2*inBuffer->capacity);
		
		
		//do we need to malloc again (?)
		hipMemcpy(outBuffer_d, outBuffer, sizeof(COOMatrix), hipMemcpyHostToDevice);
		hipMemcpy(out_rowIdxs_d, outBuffer->rowIdxs, outBuffer->capacity * sizeof(unsigned int), hipMemcpyHostToDevice);
		hipMemcpy(out_colIdxs_d, outBuffer->colIdxs, outBuffer->capacity * sizeof(unsigned int), hipMemcpyHostToDevice);
		hipMemcpy(out_values_d, outBuffer->values, outBuffer->capacity * sizeof(float), hipMemcpyHostToDevice);
		hipMemcpy(&(outBuffer_d->rowIdxs), &out_rowIdxs_d, sizeof(unsigned int*), hipMemcpyHostToDevice);
		hipMemcpy(&(outBuffer_d->colIdxs), &out_colIdxs_d, sizeof(unsigned int*), hipMemcpyHostToDevice);
		hipMemcpy(&(outBuffer_d->values), &out_values_d, sizeof(float*), hipMemcpyHostToDevice);
		
		hipFree(w_colPtrs_d);
		hipFree(w_rowIdxs_d);
		hipFree(w_values_d);
		hipFree(W_d);
	}
	
	// Copy data from GPU
    startTime(&timer);
	
	hipMemcpy(inBuffer, inBuffer_d, sizeof(CSRMatrix), hipMemcpyDeviceToHost);
	//struct fields as variables(?)
	hipMemcpy(inBuffer->rowPtrs, in_rowPtrs_d, (inBuffer->numRows + 1) * sizeof(unsigned int), hipMemcpyDeviceToHost);
	hipMemcpy(inBuffer->colIdxs, in_colIdxs_d, inBuffer->numCols * sizeof(unsigned int), hipMemcpyDeviceToHost);
	hipMemcpy(inBuffer->values, in_values_d, inBuffer->numCols * sizeof(float), hipMemcpyDeviceToHost);
	//copy pointers back (??)
	hipMemcpy(&in_rowPtrs_d, &(inBuffer_d->rowPtrs), sizeof(unsigned int*), hipMemcpyDeviceToHost);
	hipMemcpy(&in_colIdxs_d, &(inBuffer_d->colIdxs), sizeof(unsigned int*), hipMemcpyDeviceToHost);
	hipMemcpy(&in_values_d, &(inBuffer_d->values), sizeof(float*), hipMemcpyDeviceToHost);
	
	
	
	hipDeviceSynchronize();
    stopTime(&timer);
    printElapsedTime(timer, "Copy from GPU time");
	
	
	// Find nonzero rows
    startTime(&timer);
    findNonzeroRows(result, inBuffer);
	stopTimeAndPrint(&timer, "Find nonzero rows");

    // Free GPU memory
        startTime(&timer);

        hipFree(in_rowPtrs_d);
        hipFree(in_colIdxs_d);
        hipFree(in_values_d);
		hipFree(inBuffer_d);
        hipFree(out_rowIdxs_d);
        hipFree(out_colIdxs_d);
        hipFree(out_values_d);
        hipFree(outBuffer);
        
        hipDeviceSynchronize();
        stopTime(&timer);

        printElapsedTime(timer, "Deallocation time");

        // Free buffers
        startTime(&timer);
        freeCSR(Y0);
        for (unsigned int layer = 0; layer < numLayers; ++layer) {
                freeCSC(W[layer]);
        }

        stopTimeAndPrint(&timer, "Deallocate memory");
}
