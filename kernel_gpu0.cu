#include "hip/hip_runtime.h"

#include <stdio.h>

#include "kernel.h"
#include "matrix.h"
#include "timer.h"


__global__ void spmspm(COOMatrix *result){//, unsigned int* nnz_out){ 
	
	result->rowIdxs[0] = 1;
	result->colIdxs[0] = 1;
	result->values[0] = 5;
	result->nnz = 10;
	//*nnz_out = 42;
}

void sparseNN(Vector* result, COOMatrix* outBuffer, COOMatrix** layerWeights, float bias, unsigned int numLayers) {

    //outBuffer_d allocation
	COOMatrix *outBuffer_d; 
	unsigned int* out_rowIdxs_d;
	unsigned int* out_colIdxs_d;
	float* out_values_d;
	outBuffer_d->nnz = outBuffer->nnz;
	//unsigned int* out_nnz_d;
	//unsigned int* out_nnz_h = (unsigned int*) malloc(sizeof(unsigned int*));
	//*out_nnz_h = outBuffer->nnz;
	hipMalloc((void**)&outBuffer_d, sizeof(COOMatrix));
	hipMalloc((void**)&out_rowIdxs_d, outBuffer->capacity * sizeof(unsigned int));
	hipMalloc((void**)&out_colIdxs_d, outBuffer->capacity * sizeof(unsigned int));
	hipMalloc((void**)&out_values_d, outBuffer->capacity * sizeof(float));
	//hipMalloc((void**)&out_nnz_d, sizeof(unsigned int));



	//copying outbuffer
	hipMemcpy(outBuffer_d, outBuffer, sizeof(COOMatrix), hipMemcpyHostToDevice);
	hipMemcpy(out_rowIdxs_d, outBuffer->rowIdxs, outBuffer->capacity * sizeof(unsigned int), hipMemcpyHostToDevice);
	hipMemcpy(out_colIdxs_d, outBuffer->colIdxs, outBuffer->capacity * sizeof(unsigned int), hipMemcpyHostToDevice);
	hipMemcpy(out_values_d, outBuffer->values, outBuffer->capacity * sizeof(float), hipMemcpyHostToDevice);
	//hipMemcpy(out_nnz_d, out_nnz_h, sizeof(unsigned int), hipMemcpyHostToDevice);
	//hipMemcpy(out_nnz_d, &(outBuffer->nnz), sizeof(unsigned int), hipMemcpyHostToDevice);
	hipMemcpy(&(outBuffer_d->rowIdxs), &out_rowIdxs_d, sizeof(unsigned int*), hipMemcpyHostToDevice);
	hipMemcpy(&(outBuffer_d->colIdxs), &out_colIdxs_d, sizeof(unsigned int*), hipMemcpyHostToDevice);
	hipMemcpy(&(outBuffer_d->values), &out_values_d, sizeof(float*), hipMemcpyHostToDevice);
	//hipMemcpy(&(outBuffer->nnz), &out_nnz_d, sizeof(unsigned int), hipMemcpyHostToDevice);
	//hipMemcpy(&out_nnz_d, &out_nnz_h, sizeof(unsigned int*), hipMemcpyHostToDevice);
	hipDeviceSynchronize();
	printf("nnz before kernel call %d \n", outBuffer->nnz);

	spmspm <<<1, 1>>> (outBuffer_d);//, out_nnz_d);
	hipDeviceSynchronize();

	//copy back       
	hipMemcpy(outBuffer->rowIdxs, out_rowIdxs_d, outBuffer->capacity * sizeof(unsigned int), hipMemcpyDeviceToHost);
	hipMemcpy(outBuffer->colIdxs, out_colIdxs_d, outBuffer->capacity * sizeof(unsigned int), hipMemcpyDeviceToHost);
	hipMemcpy(outBuffer->values, out_values_d, outBuffer->capacity * sizeof(float), hipMemcpyDeviceToHost);
	outBuffer->nnz = outBuffer_d->nnz;
	//hipMemcpy(outBuffer->nnz, &out_nnz_d, sizeof(unsigned int), hipMemcpyDeviceToHost);
	//hipMemcpy(&(outBuffer->nnz), out_nnz_d, sizeof(unsigned int), hipMemcpyDeviceToHost);
	//hipMemcpy(out_nnz_h, out_nnz_d, sizeof(unsigned int), hipMemcpyDeviceToHost);
	
	hipDeviceSynchronize();
	printf("%f \n", outBuffer->values[0]);
	printf("nnz after kernel call %d \n", outBuffer->nnz);



}
