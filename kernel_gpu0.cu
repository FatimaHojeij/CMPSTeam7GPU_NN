#include "hip/hip_runtime.h"
// row+1; swapping; nnzidx; syncthreads
#include <stdio.h>

#include "kernel.h"
#include "matrix.h"
#include "timer.h"


#define THRESHOLD 0.000001
#define YMAX 32
#define threads 512


__global__ void spmspm(COOMatrix *result, CSRMatrix A, CSCMatrix B, float bias) {
    unsigned int r= blockIdx.y*blockDim.y +threadIdx.y;
    unsigned int c= blockIdx.x*blockDim.x + threadIdx.x;
    unsigned int rowPtrA;
    unsigned int nnzA;
        if(r < A.numRows && c < B.numCols){
                rowPtrA = A.rowPtrs[r];
                nnzA = A.rowPtrs[r + 1] - rowPtrA;
                if(nnzA>0) { // if a row is not all zeros , we do computation otherwise we skip row
                        //ptrs to cols and vals of A[r]
                        unsigned int* colIdxsA = A.colIdxs + rowPtrA;
                        float* valueA = A.values + rowPtrA;
                        //we will take one column of B
                        unsigned int colPtrB = B.colPtrs[c];
                        unsigned int nnzB = B.colPtrs[c + 1] - colPtrB;
                        if(nnzB>0) { // if a col in B is not all zeros, we do computation otherwise skip//ptrs to rows and vals of B[c]
                                unsigned int* rowIdxsB = B.rowIdxs + colPtrB;
                                float* valueB = B.values + colPtrB;
                                // Loop and find intersection
                                float sum = 0.0f;
                                unsigned int ia = 0, ib = 0;
                                while(ia < nnzA && ib < nnzB) { // loops over all non zeros from A and B and stop when there is no more non zero
                                        unsigned int colIdx = colIdxsA[ia]; //single item col index from A
                                        unsigned int rowIdx = rowIdxsB[ib]; //single item row index from B
                                        if(colIdx < rowIdx) {
                                                ia++;
                                        } else if(colIdx > rowIdx) {
                                                ib++;
                                        } else {
                                                sum += valueA[ia]*valueB[ib];// do the multiplication of the row that matches the column
                                                ia++;
                                                ib++;
                                        }
                                }
                                if(sum > THRESHOLD || sum < -THRESHOLD) { //if not smaller than abs(threshold)
                                        sum += bias; //add to it the bias
                                        //Remove negative and zero values
                                        if(sum > 0) {//if end result is positive otherwise I also do not want to add it to result
                                                if(sum>YMAX) { //make sure it is on an upper limit
                                                        sum = YMAX;
                                                }
                                                unsigned int nnzIndxTemp = atomicAdd(&(result->nnz),1); //counts how many non zero elements I have
                                                result->rowIdxs[nnzIndxTemp] = r;
                                                result->colIdxs[nnzIndxTemp] = c;
                                                result->values[nnzIndxTemp] = sum;
                                        }
                                }
                        }

                }
        }
}

COOMatrix* sortCOO(COOMatrix *A){

        // sorting rows
         for (unsigned int i = 0; i < A->nnz; i++)
                for (unsigned int j = 0; j < A->nnz-i-1; j++)
                {    if (A->rowIdxs[j] > A->rowIdxs[j+1]){
                                unsigned int r = A->rowIdxs[j];
                                unsigned int c =  A->colIdxs[j];
                                float v = A->values[j];
                                A->rowIdxs[j] = A->rowIdxs[j+1];
                                A->colIdxs[j] = A->colIdxs[j+1];
                                A->values[j] = A->values[j+1];
                                A->rowIdxs[j+1] = r;
                                A->colIdxs[j+1] = c;
                                A->values[j+1] = v;
                        }
                }

         // sorting the col
        // int count = 0;
         int begin = 0;
         for(unsigned int i  = 0 ;  i < A->nnz -1 ; i++)
         {
                 //count++;
                 if(A->rowIdxs[i] != A->rowIdxs[i+1])
                 {
                         //sort the col
                        for(int k = begin ;  k< i + begin; k++)
                                for (int m = begin ; m < i + begin - k -1 ;m++)
                                        if(A->colIdxs[m] > A->colIdxs[m+1]){
                                                unsigned int c = A->colIdxs[m];
                                                float v = A->values[m];
                                                A->colIdxs[m] = A->colIdxs[m+1];
                                                A->values[m] = A->values[m+1];
                                                A->colIdxs[m+1] =c;
                                                A->values[m+1] = v;

                                        }

                        // count = 0;
                        begin= i+1;
                }


        }
        return A;



 }
//converts from CSRMatrix to Vector and a vector of indices where the row is not all zeros
void findNonzeroRows(Vector* v, CSRMatrix* A) {
        unsigned int nnz = 0;
        for (unsigned int r = 0; r < A->numRows; ++r) {
                unsigned int rowPtrA = A->rowPtrs[r];
                unsigned int nnzA = A->rowPtrs[r + 1] - rowPtrA;
                if (nnzA > 0) {
                        if (nnz >= v->capacity) {
                                expandVectorCapacity(v, 2 * v->capacity);
                        }
                        v->data[nnz] = r;
                        ++nnz;
                }
        }
        v->nnz = nnz;
}

COOMatrix* createEmptyCOO(unsigned int numRows, unsigned int numCols, unsigned int capacity) {
        COOMatrix *coo = (COOMatrix *)malloc(sizeof(COOMatrix));
        coo->rowIdxs= (unsigned int *)malloc(capacity * sizeof(unsigned int));
        coo->colIdxs= (unsigned int *)malloc(capacity * sizeof(unsigned int));
        coo->values= (float *)malloc( capacity * sizeof(float));
        coo->numRows = numRows;
        coo->numCols = numCols;
        coo->nnz = 0;
        coo->capacity = capacity;
        return coo;
}
void sparseNN(Vector* result, COOMatrix* featureVectors, COOMatrix** layerWeights, float bias, unsigned int numLayers) {

        Timer timer;

        // Convert featureVectors to CSR
        startTime(&timer);
		CSRMatrix* Y0 = createCSRfromCOO(featureVectors);
		stopTimeAndPrint(&timer, "Convert feature vectors to CSR");

	// Convert layer weights to CSC
	startTime(&timer);
	CSCMatrix* W[numLayers];
	for (unsigned int layer = 0; layer < numLayers; ++layer) {
			W[layer] = createCSCfromCOO(layerWeights[layer]);
	}
	stopTimeAndPrint(&timer, "Convert weights to CSC");

	// Double buffers
	startTime(&timer);
	COOMatrix *tmp = createEmptyCOO(Y0->numRows, Y0->numCols, 5 * Y0->nnz);
	CSRMatrix *inBuffer = Y0;
	COOMatrix *outBuffer = tmp;
	stopTimeAndPrint(&timer, "Allocate temporary buffer");



        // Allocate GPU memory
        startTime(&timer);

        //result_d allocation
        //Vector *result_d;
        //result_d->nnz = result->nnz;
        //result_d->capacity = result->capacity;
        //hipMalloc((void**)&result_d->data, result->capacity * sizeof(unsigned int));
     //inBuffer_d allocation
        CSRMatrix inBuffer_d;
        inBuffer_d.numRows = inBuffer->numRows;
        inBuffer_d.numCols = inBuffer->numCols;
        inBuffer_d.nnz = inBuffer->nnz;
        inBuffer_d.capacity = inBuffer->capacity;
		//hipMalloc((void**)&inBuffer_d, sizeof(CSRMatrix));
        hipMalloc((void**) &inBuffer_d.rowPtrs, (inBuffer_d.numRows + 1) * sizeof(unsigned int));
        hipMalloc((void**) &inBuffer_d.colIdxs, inBuffer_d.numCols * sizeof(unsigned int));
        hipMalloc((void**) &inBuffer_d.values, inBuffer_d.numCols * sizeof(float));



        //outBuffer_d allocation
        COOMatrix *outBuffer_d;
		outBuffer_d = outBuffer;
        outBuffer_d->numRows = outBuffer->numRows;
        outBuffer_d->numCols = outBuffer->numCols;
        outBuffer_d->nnz = outBuffer->nnz;
        outBuffer_d->capacity = outBuffer->capacity;
		//hipMalloc((void**)&outBuffer_d, sizeof(COOMatrix));
        hipMalloc((void**)&outBuffer_d->rowIdxs, outBuffer->capacity * sizeof(unsigned int));
        hipMalloc((void**)&outBuffer_d->colIdxs, outBuffer->capacity * sizeof(unsigned int));
        hipMalloc((void**)&outBuffer_d->values, outBuffer->capacity * sizeof(float));

        // allocating W_d
        CSCMatrix* W_d[numLayers];
		//hipMalloc((void**)&W_d, numLayers * sizeof(CSCMatrix));
        for (unsigned int layer = 0; layer < numLayers; ++layer) {
                W_d[layer]->numRows = W[layer]->numRows;
                W_d[layer]->numCols = W[layer]->numCols;
                W_d[layer]->nnz = W[layer]->nnz;
                W_d[layer]->capacity = W[layer]->capacity;
				//hipMalloc((void**)&W_d[layer], sizeof(CSCMatrix));
                hipMalloc((void**)&W_d[layer].colPtrs, W[layer].numCols * sizeof(unsigned int));
                hipMalloc((void**)&W_d[layer].rowIdxs, W[layer].numRows * sizeof(unsigned int));
                hipMalloc((void**)&W_d[layer].values, W[layer].numRows * sizeof(float));
        }

        hipDeviceSynchronize();
        stopTime(&timer);
        printElapsedTime(timer, "Allocation time on GPU Memory");

        // Copy data to GPU
        startTime(&timer);

        //for result
        //hipMemcpy(result_d->data, result->data, result_d->capacity * sizeof(unsigned int), hipMemcpyHostToDevice);


        //for inbuffer
        hipMemcpy(inBuffer_d.rowPtrs, inBuffer->rowPtrs, inBuffer_d.numRows * sizeof(unsigned int), hipMemcpyHostToDevice);
        hipMemcpy(inBuffer_d.colIdxs, inBuffer->colIdxs, inBuffer_d.numCols * sizeof(unsigned int), hipMemcpyHostToDevice);
        hipMemcpy(inBuffer_d.values, inBuffer->values, inBuffer_d.numCols * sizeof(float), hipMemcpyHostToDevice);
        printElapsedTime(timer, "For inBuffer");
        //for outbuffer
        //hipMemcpy(outBuffer_d, outBuffer, sizeof(COOMatrix), hipMemcpyHostToDevice);
		hipMemcpy(outBuffer_d->rowIdxs, outBuffer->rowIdxs, outBuffer_d->capacity * sizeof(unsigned int), hipMemcpyHostToDevice);
        hipMemcpy(outBuffer_d->colIdxs, outBuffer->colIdxs, outBuffer_d->capacity * sizeof(unsigned int), hipMemcpyHostToDevice);
        hipMemcpy(outBuffer_d->values, outBuffer->values, outBuffer_d->capacity * sizeof(float), hipMemcpyHostToDevice);
        printElapsedTime(timer, "For outBuffer");
        //for Weights
        for (unsigned int layer = 0; layer < numLayers; ++layer) {
                hipMemcpy(W_d[layer].colPtrs, W[layer]->colPtrs, W_d[layer].numCols * sizeof(unsigned int), hipMemcpyHostToDevice);
                hipMemcpy(W_d[layer].rowIdxs, W[layer]->rowIdxs, W_d[layer].numRows * sizeof(unsigned int), hipMemcpyHostToDevice);
                hipMemcpy(W_d[layer].values, W[layer]->values, W_d[layer].numRows * sizeof(float), hipMemcpyHostToDevice);
        }

        hipDeviceSynchronize();
        stopTime(&timer);
        printElapsedTime(timer, "Copy to GPU time");

        //kernel loop
/////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
        // Loop over layers
        for (unsigned int layer = 0; layer < numLayers; ++layer) {

                // SpMSpM
                printf("Computing layer %u (SpMSpM)", layer);
                startTime(&timer);
                //unsigned int nnzIdx=0;

                //do kernel call instead
                //int outputSize = inBuffer_d->numRows * W_d[layer]->numCols;

                dim3 numThreadsPerBlock(threads, threads);
                dim3 numBlocks((W_d[layer].numCols + numThreadsPerBlock.x - 1)/numThreadsPerBlock.x,(inBuffer_d.numRows + numThreadsPerBlock.y - 1)/numThreadsPerBlock.y);
                //int numBlocks = (outputSize + numThreadsPerBlock - 1)/numThreadsPerBlock ;
                spmspm <<<numBlocks, numThreadsPerBlock>>> (outBuffer_d, inBuffer_d, W_d[layer], bias);
                hipDeviceSynchronize();
                stopTimeAndPrint(&timer, "");

                stopTimeAndPrint(&timer, "For Out Buffer");
				outBuffer = outBuffer_d;
                outBuffer->numRows = outBuffer_d.numRows ;
                outBuffer->numCols = outBuffer_d.numCols ;
                outBuffer->nnz = outBuffer_d.nnz;
                stopTimeAndPrint(&timer, "For Out Buffer MemCopy");
                hipMemcpy(outBuffer, outBuffer_d, sizeof(COOMatrix), hipMemcpyDeviceToHost);
				hipMemcpy(outBuffer->rowIdxs, outBuffer_d->rowIdxs, outBuffer_d->capacity * sizeof(unsigned int), hipMemcpyDeviceToHost);
                hipMemcpy(outBuffer->colIdxs, outBuffer_d->colIdxs, outBuffer_d->capacity * sizeof(unsigned int), hipMemcpyDeviceToHost);
                hipMemcpy(outBuffer->values, outBuffer_d->values, outBuffer_d->capacity * sizeof(float), hipMemcpyDeviceToHost);

                stopTimeAndPrint(&timer, "For Sort");
                inBuffer = createCSRfromCOO(sortCOO(outBuffer));
                stopTimeAndPrint(&timer, "Out of sort");

                inBuffer_d.numRows = inBuffer->numRows;
                inBuffer_d.numCols = inBuffer->numCols;
                inBuffer_d.nnz = inBuffer->nnz;
                inBuffer_d.capacity = inBuffer->capacity;
                hipMemcpy(inBuffer_d.rowPtrs, inBuffer->rowPtrs, inBuffer_d.numRows * sizeof(unsigned int), hipMemcpyHostToDevice);

                hipMemcpy(inBuffer_d.colIdxs, inBuffer->colIdxs, inBuffer_d.numCols * sizeof(unsigned int), hipMemcpyHostToDevice);
                hipMemcpy(inBuffer_d.values, inBuffer->values, inBuffer_d.numCols * sizeof(float), hipMemcpyHostToDevice);


                outBuffer = createEmptyCOO(inBuffer->numRows, inBuffer->numCols, 2*inBuffer->capacity);
                outBuffer_d = outBuffer;
				outBuffer_d->numRows = outBuffer->numRows;
                outBuffer_d->numCols = outBuffer->numCols;
                outBuffer_d->nnz = outBuffer->nnz;
                outBuffer_d->capacity = outBuffer->capacity;
				//hipMemcpy(outBuffer_d , outBuffer, sizeof(COOMatrix), hipMemcpyHostToDevice);
                hipMemcpy(outBuffer_d->rowIdxs, outBuffer->rowIdxs, outBuffer_d->capacity * sizeof(unsigned int), hipMemcpyHostToDevice);
                hipMemcpy(outBuffer_d->colIdxs, outBuffer->colIdxs, outBuffer_d->capacity * sizeof(unsigned int), hipMemcpyHostToDevice);
                hipMemcpy(outBuffer_d->values, outBuffer->values, outBuffer_d->capacity * sizeof(float), hipMemcpyHostToDevice);

        }

/////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
/////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
/////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
/////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////



        // Copy data from GPU
        startTime(&timer);

        // TODO

        hipMemcpy(inBuffer->rowPtrs, inBuffer_d.rowPtrs, inBuffer_d.numRows * sizeof(unsigned int), hipMemcpyDeviceToHost);
        hipMemcpy(inBuffer->colIdxs, inBuffer_d.colIdxs, inBuffer_d.numCols * sizeof(unsigned int), hipMemcpyDeviceToHost);
        hipMemcpy(inBuffer->values, inBuffer_d.values, inBuffer_d.numCols * sizeof(float), hipMemcpyDeviceToHost);


        hipDeviceSynchronize();
        stopTime(&timer);
        printElapsedTime(timer, "Copy from GPU time");

        //CPU
        // Find nonzero rows
        startTime(&timer);
        findNonzeroRows(result, inBuffer);
		stopTimeAndPrint(&timer, "Find nonzero rows");

        // Free GPU memory
        startTime(&timer);

        hipFree(inBuffer_d.rowPtrs);
        hipFree(inBuffer_d.colIdxs);
        hipFree(inBuffer_d.values);
        hipFree(outBuffer_d->rowIdxs);
        hipFree(outBuffer_d->colIdxs);
        hipFree(outBuffer_d->values);
        for (unsigned int layer = 0; layer < numLayers; ++layer) {
                hipFree(W_d[layer].colPtrs);
                hipFree(W_d[layer].rowIdxs);
                hipFree(W_d[layer].values);

        }


        hipDeviceSynchronize();
        stopTime(&timer);

        printElapsedTime(timer, "Deallocation time");

        // Free buffers
        startTime(&timer);
        freeCSR(Y0);
        for (unsigned int layer = 0; layer < numLayers; ++layer) {
                freeCSC(W[layer]);
        }

        stopTimeAndPrint(&timer, "Deallocate memory");


}
