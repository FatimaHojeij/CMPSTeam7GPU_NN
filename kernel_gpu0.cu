#include "hip/hip_runtime.h"

// row+1; swapping; nnzidx; syncthreads
#include <stdio.h>

#include "kernel.h"
#include "matrix.h"
#include "timer.h"


__global__ void spmspm(COOMatrix *result){ 
	
	result->rowIdxs[0] = 1;
	result->colIdxs[0] = 1;
	result->values[0] = 5;
	result->nnz = 10;
}

void sparseNN(Vector* result, COOMatrix* outBuffer, COOMatrix** layerWeights, float bias, unsigned int numLayers) {

    //outBuffer_d allocation
	COOMatrix *outBuffer_d; 
	unsigned int* out_rowIdxs_d;
	unsigned int* out_colIdxs_d;
	float* out_values_d;
	hipMalloc((void**)&outBuffer_d, sizeof(COOMatrix));
	hipMalloc((void**)&out_rowIdxs_d, outBuffer->capacity * sizeof(unsigned int));
	hipMalloc((void**)&out_colIdxs_d, outBuffer->capacity * sizeof(unsigned int));
	hipMalloc((void**)&out_values_d, outBuffer->capacity * sizeof(float));



	//copying outbuffer
	hipMemcpy(outBuffer_d, outBuffer, sizeof(COOMatrix), hipMemcpyHostToDevice);
	hipMemcpy(out_rowIdxs_d, outBuffer->rowIdxs, outBuffer->capacity * sizeof(unsigned int), hipMemcpyHostToDevice);
	hipMemcpy(out_colIdxs_d, outBuffer->colIdxs, outBuffer->capacity * sizeof(unsigned int), hipMemcpyHostToDevice);
	hipMemcpy(out_values_d, outBuffer->values, outBuffer->capacity * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(&(outBuffer_d->rowIdxs), &out_rowIdxs_d, sizeof(unsigned int*), hipMemcpyHostToDevice);
	hipMemcpy(&(outBuffer_d->colIdxs), &out_colIdxs_d, sizeof(unsigned int*), hipMemcpyHostToDevice);
	hipMemcpy(&(outBuffer_d->values), &out_values_d, sizeof(float*), hipMemcpyHostToDevice);
	hipDeviceSynchronize();
	printf("nnz before kernel call %d \n", outBuffer->nnz);

	spmspm <<<1, 1>>> (outBuffer_d);


	//copy back       
	hipMemcpy(outBuffer->rowIdxs, out_rowIdxs_d, outBuffer->capacity * sizeof(unsigned int), hipMemcpyDeviceToHost);
	hipMemcpy(outBuffer->colIdxs, out_colIdxs_d, outBuffer->capacity * sizeof(unsigned int), hipMemcpyDeviceToHost);
	hipMemcpy(outBuffer->values, out_values_d, outBuffer->capacity * sizeof(float), hipMemcpyDeviceToHost);

	printf("%f \n", outBuffer->values[0]);
	printf("nnz after kernel call %d \n", outBuffer->nnz);



}
