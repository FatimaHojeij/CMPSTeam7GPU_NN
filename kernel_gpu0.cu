#include "hip/hip_runtime.h"
/*#include "hip/hip_runtime.h"
#include ""

#include <stdlib.h>*/


// row+1; swapping; nnzidx; syncthreads
#include <stdio.h>

#include "kernel.h"
#include "matrix.h"
#include "timer.h"

#include<string.h>
#define THRESHOLD 0.000001
#define YMAX 32
#define threads 32
#define BLOCK_DIM 1024
#define CAPACITY 25498020

//__constant__ unsigned int u_Max;

__global__ void spmspm(COOMatrix *result, CSRMatrix A, CSCMatrix B, float bias, unsigned int* nnz_out) {
	unsigned int r = blockIdx.y*blockDim.y + threadIdx.y;
	unsigned int c = blockIdx.x*blockDim.x + threadIdx.x;


	if (r < A.numRows && c < B.numCols) {
		unsigned int rowPtrA = A.rowPtrs[r];
		unsigned int nnzA = A.rowPtrs[r + 1] - rowPtrA;

		unsigned int colPtrB = B.colPtrs[c];
		unsigned int nnzB = B.colPtrs[c + 1] - colPtrB;
		if (nnzA > 0 && nnzB > 0) { // if a row is not all zeros , we do computation otherwise we skip row
				//ptrs to cols and vals of A[r]
				//unsigned int* colIdxsA = A.colIdxs + rowPtrA;
				//float* valueA = A.values + rowPtrA;
				//we will take one column of B

				 // if a col in B is not all zeros, we do computation otherwise skip//ptrs to rows and vals of B[c]
						//unsigned int* rowIdxsB = B.rowIdxs[colPtrB];
						//float* valueB = B.values[colPtrB];
						// Loop and find intersection
			float sum = 0.0f;
			unsigned int ia = 0, ib = 0;
			while (ia < nnzA && ib < nnzB) { // loops over all non zeros from A and B and stop when there is no more non zero

				unsigned int colIdx = A.colIdxs[rowPtrA + ia]; //single item col index from A
				unsigned int rowIdx = B.rowIdxs[colPtrB + ib]; //single item row index from B
				if (rowIdx < B.nnz && colIdx < A.nnz) {
					if (colIdx < rowIdx) {
						ia++;
					}
					else if (colIdx > rowIdx) {
						ib++;
					}
					else {
						sum += A.values[rowPtrA + ia] * B.values[ib + colPtrB];// do the multiplication of the row that matches the column
						ia++;
						ib++;
					}
				}

			}
			if (sum > THRESHOLD || sum < -THRESHOLD) { //if not smaller than abs(threshold)
				sum += bias; //add to it the bias
				//Remove negative and zero values
				if (sum > 0) {//if end result is positive otherwise I also do not want to add it to result
					if (sum > YMAX) { //make sure it is on an upper limit
						sum = YMAX;
					}
					unsigned int nnzIndxTemp = atomicAdd(nnz_out, 1); //counts how many non zero elements I have
					result->rowIdxs[nnzIndxTemp] = r;
					result->colIdxs[nnzIndxTemp] = c;
					result->values[nnzIndxTemp] = sum;
				}
			}


		}
	}

}


//extern __shared__ unsigned int array[];
__global__ void histogram_private_kernel(unsigned int* rowIdxs, unsigned int* rowPtrs, unsigned int nnz, unsigned int numRows) {


	unsigned int t = blockDim.x*blockIdx.x + threadIdx.x;

	// if (t < numRows + 1) {
	// 	rowPtrs[t] = 0;
	// }

	// __syncthreads();

	if (t < nnz) {
		unsigned int rIdx = rowIdxs[t];
		atomicAdd(&rowPtrs[rIdx], 1);
	}



}


__global__ void scan_kernel(unsigned int* input, unsigned int* output, unsigned int* partialSums, unsigned int N) {

	// TODO


	unsigned int segment = 2 * blockDim.x * blockIdx.x;
	unsigned int i = segment + threadIdx.x;

	__shared__ unsigned int input_s[2 * BLOCK_DIM];

	int tid = threadIdx.x;


	if (i < N)
	{
		input_s[tid] = input[i];
	}
	else
	{
		input_s[tid] = 0;
	}
	if (i + BLOCK_DIM < N)
	{
		input_s[tid + BLOCK_DIM] = input[i + BLOCK_DIM];
	}
	else
	{
		input_s[tid + BLOCK_DIM] = 0;
	}
	__syncthreads();


	//reduction step
	for (unsigned int stride = 1; stride <= BLOCK_DIM; stride *= 2)
	{
		int index = (threadIdx.x + 1) * 2 * stride - 1;
		if (index < 2 * BLOCK_DIM)
			input_s[index] += input_s[index - stride];
		__syncthreads();
	}

	//save partial sum
	if (threadIdx.x == 0)
	{
		partialSums[blockIdx.x] = input_s[2 * BLOCK_DIM - 1];
		input_s[2 * BLOCK_DIM - 1] = 0.0f;

	}

	__syncthreads();

	//post reduction step
	for (unsigned int stride = BLOCK_DIM; stride > 0; stride /= 2)
	{
		int index = (threadIdx.x + 1) * 2 * stride - 1;

		if (index < 2 * BLOCK_DIM)
		{
			//add then swap
			unsigned int temp = input_s[index];
			input_s[index] += input_s[index - stride];
			input_s[index - stride] = temp;
		}

		__syncthreads();
	}


	if (i < N)
	{
		output[i] = input_s[tid];
	}
	if (i + BLOCK_DIM < N)
	{
		output[i + BLOCK_DIM] = input_s[tid + BLOCK_DIM];
	}

}

__global__ void add_kernel(unsigned int* output, unsigned int* partialSums, unsigned int N) {

	// TODO
	unsigned int i = 2 * blockIdx.x*blockDim.x + threadIdx.x;
	if (blockIdx.x != 0) {
		if (i < N) {
			output[i] += partialSums[blockIdx.x];
		}
		if (i + BLOCK_DIM < N) {
			output[i + BLOCK_DIM] += partialSums[blockIdx.x];
		}
	}

}
//output_d rowptrs n = numrows +1
void scan_gpu_d(unsigned int* input_d, unsigned int* output_d, unsigned int N) {

	// Configurations
	const unsigned int numThreadsPerBlock = BLOCK_DIM;
	const unsigned int numElementsPerBlock = 2 * numThreadsPerBlock;
	const unsigned int numBlocks = (N + numElementsPerBlock - 1) / numElementsPerBlock;

	// Allocate partial sums

	unsigned int *partialSums_d;
	hipMalloc((void**)&partialSums_d, numBlocks * sizeof(unsigned int));
	hipDeviceSynchronize();


	scan_kernel << < numBlocks, numThreadsPerBlock >> > (input_d, output_d, partialSums_d, N);
	hipDeviceSynchronize();


	// Scan partial sums then add
	if (numBlocks > 1) {

		// Scan partial sums
		scan_gpu_d(partialSums_d, partialSums_d, numBlocks);

		// Add scanned sums
		add_kernel << < numBlocks, numThreadsPerBlock >> > (output_d, partialSums_d, N);

	}

	// Free memory

	hipFree(partialSums_d);
	hipDeviceSynchronize();

}



__global__ void Binning_kernel(unsigned int* inrowIdxs, unsigned int* incolIdxs, float* invalues, unsigned int* rowPtrs, unsigned int* colIdxs, float* values, unsigned int nnz, unsigned int numRows, unsigned int* rowPtrsBin) {

	int i = threadIdx.x + blockIdx.x*blockDim.x;


	// if (i < numRows + 1) {
	// 	rowPtrsBin[i] = 0;
	// }

	// __syncthreads();

	if (i < nnz) {
		unsigned int row = inrowIdxs[i];
		unsigned int col = incolIdxs[i];
		float val = invalues[i];
		unsigned int init = rowPtrs[row];
		unsigned int nnzIdx = atomicAdd(&rowPtrsBin[row], 1);
		colIdxs[nnzIdx+init] = col;
		values[nnzIdx+init]=val;
	}


}

__global__ void  sorting_kernel( unsigned int* colIdxs, float* values,unsigned int* rowPtrs, unsigned int numRows){
	int i = threadIdx.x + blockIdx.x*blockDim.x;

	if (i < numRows) {
		unsigned int rowPtrA = rowPtrs[i];
		unsigned int nnzA = (rowPtrs[i + 1] - rowPtrs[i]);
		if(nnzA>0)
			for (unsigned int j = rowPtrA; j < rowPtrA + nnzA - 1;++j) {

				for (unsigned int k = rowPtrA; k < rowPtrA + nnzA - j - 1; ++k) {
					if (colIdxs[k] > colIdxs[k + 1]) {
						//swap col
						unsigned int tmp = colIdxs[k];
						colIdxs[k] = colIdxs[k + 1];
						colIdxs[k + 1] = tmp;
						//swap float
						float valtmp = values[k];
						values[k] = values[k + 1];
						values[k + 1] = valtmp;
					}
				}
			}
		
	}

}

//converts from CSRMatrix to Vector and a vector of indices where the row is not all zeros
void findNonzeroRows(Vector* v, CSRMatrix* A) {
	unsigned int nnz = 0;
	for (unsigned int r = 0; r < A->numRows; ++r) {
		unsigned int rowPtrA = A->rowPtrs[r];
		unsigned int nnzA = A->rowPtrs[r + 1] - rowPtrA;
		if (nnzA > 0) {
			if (nnz >= v->capacity) {
				expandVectorCapacity(v, 2 * v->capacity);
			}
			v->data[nnz] = r;
			++nnz;
		}
	}
	v->nnz = nnz;
}

COOMatrix* createEmptyCOO(unsigned int numRows, unsigned int numCols, unsigned int capacity) {
	COOMatrix *coo = (COOMatrix *)malloc(sizeof(COOMatrix));
	coo->rowIdxs = (unsigned int *)malloc(capacity * sizeof(unsigned int));
	coo->colIdxs = (unsigned int *)malloc(capacity * sizeof(unsigned int));
	coo->values = (float *)malloc(capacity * sizeof(float));
	coo->numRows = numRows;
	coo->numCols = numCols;
	coo->nnz = 0;
	coo->capacity = CAPACITY;
	for (unsigned int i = 0; i < coo->capacity;++i) {
		coo->rowIdxs[i] = 0;
		coo->colIdxs[i] = 0;
		coo->values[i] = 0.0f;
	}
	return coo;
}

void sparseNN(Vector* result, COOMatrix* featureVectors, COOMatrix** layerWeights, float bias, unsigned int numLayers) {
	//const unsigned int _numLayers = 120;

	Timer timer;

	// Convert featureVectors to CSR
	startTime(&timer);
	CSRMatrix* Y0 = createCSRfromCOO(featureVectors);
	stopTimeAndPrint(&timer, "Convert feature vectors to CSR");

	// Convert layer weights to CSC
	startTime(&timer);
	CSCMatrix* W[numLayers];
	//CSCMatrix* W[_numLayers];
	for (unsigned int layer = 0; layer < numLayers; ++layer) {
		W[layer] = createCSCfromCOO(layerWeights[layer]);
	}
	stopTimeAndPrint(&timer, "Convert weights to CSC");

	// Double buffers
	startTime(&timer);
	COOMatrix *tmp = createEmptyCOO(Y0->numRows, Y0->numCols, CAPACITY);
	CSRMatrix *inBuffer = Y0;
	COOMatrix *outBuffer = tmp;
	stopTimeAndPrint(&timer, "Allocate temporary buffer");

	inBuffer->capacity = CAPACITY;

	// Allocate GPU memory
	startTime(&timer);

	outBuffer->capacity = CAPACITY;

	//allocating inbuffer address and value
	CSRMatrix inBuffer_d;
	//CSRMatrix* inBuffer_d;
	inBuffer_d.numRows = inBuffer->numRows;
	inBuffer_d.numCols = inBuffer->numCols;
	inBuffer_d.nnz = inBuffer->nnz;
	inBuffer_d.capacity = inBuffer->capacity;
	hipMalloc((void**)&inBuffer_d.rowPtrs, (inBuffer->numRows + 1) * sizeof(unsigned int));
	hipMalloc((void**)&inBuffer_d.colIdxs, CAPACITY * sizeof(unsigned int));
	hipMalloc((void**)&inBuffer_d.values, CAPACITY * sizeof(float));

	hipMemcpy(inBuffer_d.rowPtrs, inBuffer->rowPtrs, (inBuffer->numRows + 1) * sizeof(unsigned int), hipMemcpyHostToDevice);
	hipMemcpy(inBuffer_d.colIdxs, inBuffer->colIdxs, (inBuffer->nnz) * sizeof(unsigned int), hipMemcpyHostToDevice);
	hipMemcpy(inBuffer_d.values, inBuffer->values, inBuffer->nnz * sizeof(float), hipMemcpyHostToDevice);

	//hipMalloc(&inBuffer_d, sizeof(CSRMatrix));

	//hipMemcpy(inBuffer_d,&inBuffer_d,sizeof(CSRMatrix),hipMemcpyHostToDevice);

	printf("inbuffer allocated\n");

	/////////////////////////

	//outBuffer_d allocation
	COOMatrix *outBuffer_d;
	unsigned int* out_rowIdxs_d;
	unsigned int* out_colIdxs_d;
	float* out_values_d;
	unsigned int* out_nnz_d;
	unsigned int* out_nnz_h = (unsigned int*)malloc(sizeof(unsigned int*));
	*out_nnz_h = outBuffer->nnz;
	hipMalloc((void**)&outBuffer_d, sizeof(COOMatrix));
	hipMalloc((void**)&out_rowIdxs_d, outBuffer->capacity * sizeof(unsigned int));
	hipMalloc((void**)&out_colIdxs_d, outBuffer->capacity * sizeof(unsigned int));
	hipMalloc((void**)&out_values_d, outBuffer->capacity * sizeof(float));
	hipMalloc((void**)&out_nnz_d, sizeof(unsigned int));

	//copying outbuffer
	hipMemcpy(outBuffer_d, outBuffer, sizeof(COOMatrix), hipMemcpyHostToDevice);
	hipMemcpy(out_rowIdxs_d, outBuffer->rowIdxs, outBuffer->capacity * sizeof(unsigned int), hipMemcpyHostToDevice);
	hipMemcpy(out_colIdxs_d, outBuffer->colIdxs, outBuffer->capacity * sizeof(unsigned int), hipMemcpyHostToDevice);
	hipMemcpy(out_values_d, outBuffer->values, outBuffer->capacity * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(out_nnz_d, out_nnz_h, sizeof(unsigned int), hipMemcpyHostToDevice);
	hipMemcpy(&(outBuffer_d->rowIdxs), &out_rowIdxs_d, sizeof(unsigned int*), hipMemcpyHostToDevice);
	hipMemcpy(&(outBuffer_d->colIdxs), &out_colIdxs_d, sizeof(unsigned int*), hipMemcpyHostToDevice);
	hipMemcpy(&(outBuffer_d->values), &out_values_d, sizeof(float*), hipMemcpyHostToDevice);
	hipMemcpy(&(outBuffer_d->numRows), &(outBuffer->numRows), sizeof(unsigned int), hipMemcpyHostToDevice);

	hipDeviceSynchronize();



	printf("outbuffer allocated\n");
	//////////////////////////////////


	// allocating W_d
	//CSCMatrix W_d[_numLayers];
	CSCMatrix W_d[numLayers];
	for (unsigned int layer = 0; layer < numLayers; ++layer) {
		W_d[layer].numRows = W[layer]->numRows;
		W_d[layer].numCols = W[layer]->numCols;
		W_d[layer].nnz = W[layer]->nnz;
		W_d[layer].capacity = W[layer]->capacity;
		hipMalloc((void**)&W_d[layer].colPtrs, (W[layer]->numCols + 1) * sizeof(unsigned int));
		hipMalloc((void**)&W_d[layer].rowIdxs, W[layer]->capacity * sizeof(unsigned int));
		hipMalloc((void**)&W_d[layer].values, W[layer]->capacity * sizeof(float));
	}

	for (unsigned int layer = 0; layer < numLayers; ++layer) {
		hipMemcpy(W_d[layer].colPtrs, W[layer]->colPtrs, (W[layer]->numCols + 1) * sizeof(unsigned int), hipMemcpyHostToDevice);
		hipMemcpy(W_d[layer].rowIdxs, W[layer]->rowIdxs, W[layer]->capacity * sizeof(unsigned int), hipMemcpyHostToDevice);
		hipMemcpy(W_d[layer].values, W[layer]->values, W[layer]->capacity * sizeof(float), hipMemcpyHostToDevice);
	}

	hipDeviceSynchronize();
	stopTime(&timer);
	printElapsedTime(timer, "Allocation and copy time on GPU Memory");



	//unsigned int uMax = (unsigned int)~0;
	//hipMemcpyToSymbol(HIP_SYMBOL(&u_Max), &uMax, sizeof(unsigned int));

	unsigned int  *rowPtrstmp_d;
	//rowPtrstmp = (unsigned int *)malloc((inBuffer_d.numRows + 1) * sizeof(unsigned int));
	unsigned int *rowPtrstmp;
	rowPtrstmp = (unsigned int *)malloc((inBuffer_d.numRows + 1) * sizeof(unsigned int));

	//memset(rowPtrstmp, 0, sizeof (unsigned int) * (inBuffer_d.numRows + 1));


	hipMalloc((void**)&rowPtrstmp_d, (inBuffer_d.numRows + 1) * sizeof(unsigned int));

	for(unsigned int i=0; i<inBuffer_d.numRows+1;i++){
	 	rowPtrstmp[i]=0;
	}

	hipMemcpy(rowPtrstmp_d, rowPtrstmp, (inBuffer_d.numRows + 1) * sizeof(unsigned int), hipMemcpyHostToDevice);

	//kernel loop
/////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
		// Loop over layers
	for (unsigned int layer = 0; layer < numLayers; ++layer) {

		// hipMemcpy(out_nnz_h, out_nnz_d, sizeof(unsigned int), hipMemcpyDeviceToHost);
		// printf(" before nnz %d\n", *out_nnz_h);
		// if(layer!=0){
		// 	hipMemcpy(inBuffer->rowPtrs, inBuffer_d.rowPtrs, (inBuffer_d.numRows + 1) * sizeof(unsigned int), hipMemcpyDeviceToHost);
		// 	hipMemcpy(inBuffer->colIdxs, inBuffer_d.colIdxs, inBuffer_d.nnz * sizeof(unsigned int), hipMemcpyDeviceToHost);
		// 	hipMemcpy(inBuffer->values, inBuffer_d.values, inBuffer_d.nnz * sizeof(float), hipMemcpyDeviceToHost);

		// 	printf("Layer %d \n",layer)
		// 	for(int i=0; i<inBuffer_d.numRows+1;++i){

		// 		unsigned int rowPtr = inBuffer->row[i];
		// 		unsigned int nnzr = inBuffer->row[i+1]-inBuffer->row[i+1];

		// 		for(unsigned int j =rowPtr;j<rowPtr+nnzr;++j){
		// 			printf("row %u , col %u, val %f\n",i,inBuffer->colIdxs[j],inBuffer->values[j])
		// 		}
		// 	}
		// }	

		// SpMSpM
		printf("Computing layer %u (SpMSpM)", layer);
		startTime(&timer);



		dim3 numThreadsPerBlock3(threads, threads);
		dim3 numBlocks3((W_d[layer].numCols + numThreadsPerBlock3.x - 1) / numThreadsPerBlock3.x, (inBuffer_d.numRows + numThreadsPerBlock3.y - 1) / numThreadsPerBlock3.y);

		spmspm << <numBlocks3, numThreadsPerBlock3 >> > (outBuffer_d, inBuffer_d, W_d[layer], bias, out_nnz_d);

		hipDeviceSynchronize();



		stopTimeAndPrint(&timer, "");

		hipMemcpy(out_nnz_h, out_nnz_d, sizeof(unsigned int), hipMemcpyDeviceToHost);
		printf("nnz %d\n", *out_nnz_h);


		inBuffer_d.nnz = *out_nnz_h;
		inBuffer_d.numCols = W_d[layer].numCols;
		inBuffer->numCols = inBuffer_d.numCols;
		inBuffer->numRows = inBuffer_d.numRows;
		inBuffer->nnz = inBuffer_d.nnz;

		hipDeviceSynchronize();

		printf("kernel time for layer %u", layer);
		stopTimeAndPrint(&timer, "");

		startTime(&timer);
		//calling histogram to fill rowPtrs of inBuffer
		unsigned int numThreadsPerBlock = 1024;
		unsigned int numBlocks = (*out_nnz_h + numThreadsPerBlock - 1) / numThreadsPerBlock;


		// hipMemcpy(outBuffer->rowIdxs, out_rowIdxs_d, outBuffer->capacity * sizeof(unsigned int), hipMemcpyDeviceToHost);
		// hipMemcpy(out_rowIdxs_d, outBuffer->rowIdxs, outBuffer->capacity * sizeof(unsigned int), hipMemcpyHostToDevice);
		if(layer!=0)
			hipMemcpy(rowPtrstmp_d, rowPtrstmp, (inBuffer_d.numRows + 1) * sizeof(unsigned int), hipMemcpyHostToDevice);
		
		histogram_private_kernel << < numBlocks, numThreadsPerBlock >> > (out_rowIdxs_d, rowPtrstmp_d, *out_nnz_h, inBuffer_d.numRows);

		hipDeviceSynchronize();

		printf("Histogram time for layer %u", layer);
		stopTimeAndPrint(&timer, "");

		startTime(&timer);

		//calling the scan kernel to scan kernel ptrs
		const unsigned int numElementsPerBlock = 2 * numThreadsPerBlock;
		numBlocks = ((inBuffer_d.numRows + 1) + numElementsPerBlock - 1) / numElementsPerBlock;

		// Allocate partial sums
		unsigned int *partialSums_d;
		hipMalloc((void**)&partialSums_d, numBlocks * sizeof(unsigned int));
		hipDeviceSynchronize();

		// Call kernel
		scan_kernel << < numBlocks, numThreadsPerBlock >> > (rowPtrstmp_d, inBuffer_d.rowPtrs, partialSums_d, inBuffer_d.numRows + 1);

		hipDeviceSynchronize();
		// Scan partial sums then add

		if (numBlocks > 1) {

			// Scan partial sums
			scan_gpu_d(partialSums_d, partialSums_d, numBlocks);

			// Add scanned sums
			add_kernel << < numBlocks, numThreadsPerBlock >> > (inBuffer_d.rowPtrs, partialSums_d, inBuffer_d.numRows + 1);

		}

		hipDeviceSynchronize();
				

		   
		hipMemcpy(rowPtrstmp, inBuffer_d.rowPtrs, sizeof(unsigned int) * (inBuffer_d.numRows + 1), hipMemcpyDeviceToHost);

		//printf("test %u\n", rowPtrstmp[inBuffer_d.numRows]);

		// Free memory

		hipFree(partialSums_d);

		printf("Scan time for layer %u", layer);
		stopTimeAndPrint(&timer, "");
		startTime(&timer);

		//Binning
		for(unsigned int i=0; i<inBuffer_d.numRows+1;i++){
			rowPtrstmp[i]=0;
		}

		hipMemcpy(rowPtrstmp_d, rowPtrstmp, (inBuffer_d.numRows + 1) * sizeof(unsigned int), hipMemcpyHostToDevice);
		hipDeviceSynchronize();
		numBlocks = (*out_nnz_h + numThreadsPerBlock - 1) / numThreadsPerBlock;

		Binning_kernel << < numBlocks, numThreadsPerBlock >> > (out_rowIdxs_d, out_colIdxs_d, out_values_d, inBuffer_d.rowPtrs, inBuffer_d.colIdxs, inBuffer_d.values, *out_nnz_h, inBuffer_d.numRows,rowPtrstmp_d);

		hipDeviceSynchronize();

		// hipFree(rowPtrstmp_d);
		// free(rowPtrstmp);
		//Sorting
		numBlocks = ((inBuffer_d.numRows +1) + numThreadsPerBlock - 1) / numThreadsPerBlock;


		sorting_kernel <<< numBlocks, numThreadsPerBlock >>>(inBuffer_d.colIdxs, inBuffer_d.values, inBuffer_d.rowPtrs, inBuffer_d.numRows);
		// hipMemcpy(outBuffer->colIdxs, inBuffer_d.colIdxs, inBuffer_d.capacity * sizeof(unsigned int), hipMemcpyDeviceToHost);

		// hipMemcpy(outBuffer->values, inBuffer_d.values, inBuffer_d.capacity * sizeof(float), hipMemcpyDeviceToHost);

		FILE* f = fopen("./out_gpu_sorting.txt","w");
		//hipMemcpy(inBuffer->rowPtrs, inBuffer_d.rowPtrs, (inBuffer_d.numRows + 1) * sizeof(unsigned int), hipMemcpyDeviceToHost);
		hipMemcpy(inBuffer->colIdxs, inBuffer_d.colIdxs, inBuffer_d.nnz * sizeof(unsigned int), hipMemcpyDeviceToHost);
		hipMemcpy(inBuffer->values, inBuffer_d.values, inBuffer_d.nnz * sizeof(float), hipMemcpyDeviceToHost);

		//printf("Layer %d \n",layer)
		for(int i=0; i<inBuffer_d.nnz;++i){
			
			fprintf(f,"col %u  val %f\n",inBuffer->colIdxs[i],inBuffer->values[i]);

		}
		fclose(f);
		break;
		hipDeviceSynchronize();

		//empty the outbuffer
		printf("Converting time for layer %u", layer);
		stopTimeAndPrint(&timer, "");

		// hipMemcpy(outBuffer_d, outBuffer, sizeof(COOMatrix), hipMemcpyHostToDevice);
		// hipMemcpy(out_rowIdxs_d, outBuffer->rowIdxs, outBuffer->capacity * sizeof(unsigned int), hipMemcpyHostToDevice);
		// hipMemcpy(out_colIdxs_d, outBuffer->colIdxs, outBuffer->capacity * sizeof(unsigned int), hipMemcpyHostToDevice);
		// hipMemcpy(out_values_d, outBuffer->values, outBuffer->capacity * sizeof(float), hipMemcpyHostToDevice);
		*out_nnz_h =0;
		hipMemcpy(out_nnz_d, out_nnz_h, sizeof(unsigned int), hipMemcpyHostToDevice);
		// hipMemcpy(&(outBuffer_d->rowIdxs), &out_rowIdxs_d, sizeof(unsigned int*), hipMemcpyHostToDevice);
		// hipMemcpy(&(outBuffer_d->colIdxs), &out_colIdxs_d, sizeof(unsigned int*), hipMemcpyHostToDevice);
		// hipMemcpy(&(outBuffer_d->values), &out_values_d, sizeof(float*), hipMemcpyHostToDevice);
		//hipMemcpy(&(outBuffer_d->numRows), &(outBuffer->numRows), sizeof(unsigned int), hipMemcpyHostToDevice);


		for(unsigned int i=0; i<inBuffer_d.numRows+1;i++){
			rowPtrstmp[i]=0;
	   	}

		hipDeviceSynchronize();

		
	}

	/////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
	/////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////


	// Copy data from GPU
	startTime(&timer);
	return;

	//TODO
	inBuffer->numRows = inBuffer_d.numRows;
	inBuffer->numCols = inBuffer_d.numCols;
	inBuffer->nnz = inBuffer_d.nnz;
	hipMemcpy(inBuffer->rowPtrs, inBuffer_d.rowPtrs, (inBuffer_d.numRows + 1) * sizeof(unsigned int), hipMemcpyDeviceToHost);
	hipMemcpy(inBuffer->colIdxs, inBuffer_d.colIdxs, inBuffer_d.nnz * sizeof(unsigned int), hipMemcpyDeviceToHost);
	hipMemcpy(inBuffer->values, inBuffer_d.values, inBuffer_d.nnz * sizeof(float), hipMemcpyDeviceToHost);


	hipDeviceSynchronize();
	stopTime(&timer);
	printElapsedTime(timer, "Copy from GPU time");

	//CPU
	// Find 
	//nonzero rows
	startTime(&timer);
	findNonzeroRows(result, inBuffer);
	stopTimeAndPrint(&timer, "Find nonzero rows");

	// Free GPU memory
	startTime(&timer);
	hipFree(inBuffer_d.rowPtrs);
	hipFree(inBuffer_d.colIdxs);
	hipFree(inBuffer_d.values);
	hipFree(outBuffer_d);


	// hipFree(tmpOutBuffer.rowIdxs);
	// hipFree(tmpOutBuffer.colIdxs);
	// hipFree(tmpOutBuffer.values);

	//hipFree(inBuffer_d);

	free(inBuffer);
	free(outBuffer);
	for (unsigned int layer = 0; layer < numLayers; ++layer) {
		hipFree(W_d[layer].colPtrs);
		hipFree(W_d[layer].rowIdxs);
		hipFree(W_d[layer].values);

	}


	hipDeviceSynchronize();
	stopTime(&timer);

	printElapsedTime(timer, "Deallocation time");

	// Free buffers
	startTime(&timer);
	//freeCSR(Y0);
	for (unsigned int layer = 0; layer < numLayers; ++layer) {
		freeCSC(W[layer]);
	}

	stopTimeAndPrint(&timer, "Deallocate memory");


}
