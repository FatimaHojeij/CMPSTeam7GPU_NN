#include "hip/hip_runtime.h"

// row+1; swapping; nnzidx; syncthreads
#include <stdio.h>

#include "kernel.h"
#include "matrix.h"
#include "timer.h"


#define THRESHOLD 0.000001
#define YMAX 32
#define threads 512


__global__ void spmspm(COOMatrix *result, CSRMatrix A, CSCMatrix B, float bias) {
    unsigned int r= blockIdx.y*blockDim.y +threadIdx.y;
    unsigned int c= blockIdx.x*blockDim.x + threadIdx.x;
    unsigned int rowPtrA;
    unsigned int nnzA;
	result->rowIdxs[0] = r;
	result->colIdxs[0] = c;
	result->values[0] = 5;
        if(r < A.numRows && c < B.numCols){
                rowPtrA = A.rowPtrs[r];
                nnzA = A.rowPtrs[r + 1] - rowPtrA;
                if(nnzA>0) { // if a row is not all zeros , we do computation otherwise we skip row
                        //ptrs to cols and vals of A[r]
                        unsigned int* colIdxsA = A.colIdxs + rowPtrA;
                        float* valueA = A.values + rowPtrA;
                        //we will take one column of B
                        unsigned int colPtrB = B.colPtrs[c];
                        unsigned int nnzB = B.colPtrs[c + 1] - colPtrB;
                        if(nnzB>0) { // if a col in B is not all zeros, we do computation otherwise skip//ptrs to rows and vals of B[c]
                                unsigned int* rowIdxsB = B.rowIdxs + colPtrB;
                                float* valueB = B.values + colPtrB;
                                // Loop and find intersection
                                float sum = 0.0f;
                                unsigned int ia = 0, ib = 0;
                                while(ia < nnzA && ib < nnzB) { // loops over all non zeros from A and B and stop when there is no more non zero
                                        unsigned int colIdx = colIdxsA[ia]; //single item col index from A
                                        unsigned int rowIdx = rowIdxsB[ib]; //single item row index from B
                                        if(colIdx < rowIdx) {
                                                ia++;
                                        } else if(colIdx > rowIdx) {
                                                ib++;
                                        } else {
                                                sum += valueA[ia]*valueB[ib];// do the multiplication of the row that matches the column
                                                ia++;
                                                ib++;
                                        }
                                }
                                if(sum > THRESHOLD || sum < -THRESHOLD) { //if not smaller than abs(threshold)
                                        sum += bias; //add to it the bias
                                        //Remove negative and zero values
                                        if(sum > 0) {//if end result is positive otherwise I also do not want to add it to result
                                                if(sum>YMAX) { //make sure it is on an upper limit
                                                        sum = YMAX;
                                                }
                                                unsigned int nnzIndxTemp = atomicAdd(&(result->nnz),1); //counts how many non zero elements I have
                                                result->rowIdxs[nnzIndxTemp] = r;
                                                result->colIdxs[nnzIndxTemp] = c;
                                                result->values[nnzIndxTemp] = 5;
                                        }
                                }
                        }

                }
        }
}

COOMatrix* sortCOO(COOMatrix *A){

        // sorting rows
         for (unsigned int i = 0; i < A->nnz; i++)
                for (unsigned int j = 0; j < A->nnz-i-1; j++)
                {    if (A->rowIdxs[j] > A->rowIdxs[j+1]){
                                unsigned int r = A->rowIdxs[j];
                                unsigned int c =  A->colIdxs[j];
                                float v = A->values[j];
                                A->rowIdxs[j] = A->rowIdxs[j+1];
                                A->colIdxs[j] = A->colIdxs[j+1];
                                A->values[j] = A->values[j+1];
                                A->rowIdxs[j+1] = r;
                                A->colIdxs[j+1] = c;
                                A->values[j+1] = v;
                        }
                }

         // sorting the col
        // int count = 0;
         int begin = 0;
         for(unsigned int i  = 0 ;  i < A->nnz -1 ; i++)
         {
                 //count++;
                 if(A->rowIdxs[i] != A->rowIdxs[i+1])
                 {
                         //sort the col
                        for(int k = begin ;  k< i + begin; k++)
                                for (int m = begin ; m < i + begin - k -1 ;m++)
                                        if(A->colIdxs[m] > A->colIdxs[m+1]){
                                                unsigned int c = A->colIdxs[m];
                                                float v = A->values[m];
                                                A->colIdxs[m] = A->colIdxs[m+1];
                                                A->values[m] = A->values[m+1];
                                                A->colIdxs[m+1] =c;
                                                A->values[m+1] = v;

                                        }

                        // count = 0;
                        begin= i+1;
                }


        }
        return A;



 }
//converts from CSRMatrix to Vector and a vector of indices where the row is not all zeros
void findNonzeroRows(Vector* v, CSRMatrix* A) {
        unsigned int nnz = 0;
        for (unsigned int r = 0; r < A->numRows; ++r) {
                unsigned int rowPtrA = A->rowPtrs[r];
                unsigned int nnzA = A->rowPtrs[r + 1] - rowPtrA;
                if (nnzA > 0) {
                        if (nnz >= v->capacity) {
                                expandVectorCapacity(v, 2 * v->capacity);
                        }
                        v->data[nnz] = r;
                        ++nnz;
                }
        }
        v->nnz = nnz;
}

COOMatrix* createEmptyCOO(unsigned int numRows, unsigned int numCols, unsigned int capacity) {
        COOMatrix *coo = (COOMatrix *)malloc(sizeof(COOMatrix));
        coo->rowIdxs= (unsigned int *)malloc(capacity * sizeof(unsigned int));
        coo->colIdxs= (unsigned int *)malloc(capacity * sizeof(unsigned int));
        coo->values= (float *)malloc( capacity * sizeof(float));
        coo->numRows = numRows;
        coo->numCols = numCols;
        coo->nnz = 0;
        coo->capacity = capacity;
        return coo;
}
void sparseNN(Vector* result, COOMatrix* featureVectors, COOMatrix** layerWeights, float bias, unsigned int numLayers) {

        Timer timer;

        // Convert featureVectors to CSR
        startTime(&timer);
 	CSRMatrix* Y0 = createCSRfromCOO(featureVectors);
	stopTimeAndPrint(&timer, "Convert feature vectors to CSR");

	// Convert layer weights to CSC
	startTime(&timer);
	CSCMatrix* W[numLayers];
	for (unsigned int layer = 0; layer < numLayers; ++layer) {
			W[layer] = createCSCfromCOO(layerWeights[layer]);
	}
	stopTimeAndPrint(&timer, "Convert weights to CSC");

	// Double buffers
	startTime(&timer);
	COOMatrix *tmp = createEmptyCOO(Y0->numRows, Y0->numCols, 5 * Y0->nnz);
	CSRMatrix *inBuffer = Y0;
	COOMatrix *outBuffer = tmp;
	stopTimeAndPrint(&timer, "Allocate temporary buffer");



        // Allocate GPU memory
        startTime(&timer);

        //result_d allocation
        //Vector *result_d;
        //result_d->nnz = result->nnz;
        //result_d->capacity = result->capacity;
        //hipMalloc((void**)&result_d->data, result->capacity * sizeof(unsigned int));
        //inBuffer_d allocation
        

        
        CSRMatrix inBuffer_d;
        unsigned int* in_rowPtrs_d;
    unsigned int* in_colIdxs_d;
    float* in_values_d;
	hipMalloc((void**) &inBuffer_d, sizeof(CSRMatrix));
    hipMalloc((void**) &in_rowPtrs_d, (inBuffer->numRows + 1) * sizeof(unsigned int));
    hipMalloc((void**) &in_colIdxs_d, inBuffer->numCols * sizeof(unsigned int));
    hipMalloc((void**) &in_values_d, inBuffer->numCols * sizeof(float));



        //outBuffer_d allocation
        COOMatrix *outBuffer_d;
	unsigned int* out_rowIdxs_d;
    unsigned int* out_colIdxs_d;
    float* out_values_d;
    hipMalloc((void**)&outBuffer_d, sizeof(COOMatrix));
    hipMalloc((void**)&out_rowIdxs_d, outBuffer->capacity * sizeof(unsigned int));
    hipMalloc((void**)&out_colIdxs_d, outBuffer->capacity * sizeof(unsigned int));
    hipMalloc((void**)&out_values_d, outBuffer->capacity * sizeof(float));
        
        printElapsedTime(timer, "For outBuffer");
        
        //checkCuda(hipMalloc(&outBuffer_d, sizeof(COOMatrix)));

        
       //copying inbuffer
	hipMemcpy(&inBuffer_d, inBuffer, sizeof(CSRMatrix), hipMemcpyHostToDevice);
	hipMemcpy(in_rowPtrs_d, inBuffer->rowPtrs, (inBuffer->numRows + 1) * sizeof(unsigned int), hipMemcpyHostToDevice);
	hipMemcpy(in_colIdxs_d, inBuffer->colIdxs, inBuffer->numCols * sizeof(unsigned int), hipMemcpyHostToDevice);
	hipMemcpy(in_values_d, inBuffer->values, inBuffer->numCols * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(&(inBuffer_d.rowPtrs), &in_rowPtrs_d, sizeof(unsigned int*), hipMemcpyHostToDevice);
	hipMemcpy(&(inBuffer_d.colIdxs), &in_colIdxs_d, sizeof(unsigned int*), hipMemcpyHostToDevice);
	hipMemcpy(&(inBuffer_d.values), &in_values_d, sizeof(float*), hipMemcpyHostToDevice);
	printElapsedTime(timer, "For inBuffer");
	
	//copying outbuffer
    hipMemcpy(outBuffer_d, outBuffer, sizeof(COOMatrix), hipMemcpyHostToDevice);
	hipMemcpy(out_rowIdxs_d, outBuffer->rowIdxs, outBuffer->capacity * sizeof(unsigned int), hipMemcpyHostToDevice);
    hipMemcpy(out_colIdxs_d, outBuffer->colIdxs, outBuffer->capacity * sizeof(unsigned int), hipMemcpyHostToDevice);
    hipMemcpy(out_values_d, outBuffer->values, outBuffer->capacity * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(&(outBuffer_d->rowIdxs), &out_rowIdxs_d, sizeof(unsigned int*), hipMemcpyHostToDevice);
	hipMemcpy(&(outBuffer_d->colIdxs), &out_colIdxs_d, sizeof(unsigned int*), hipMemcpyHostToDevice);
	hipMemcpy(&(outBuffer_d->values), &out_values_d, sizeof(float*), hipMemcpyHostToDevice);
    printElapsedTime(timer, "For outBuffer");
        hipDeviceSynchronize();
        stopTime(&timer);
        printElapsedTime(timer, "Allocation time on GPU Memory");


        //kernel loop
/////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
        // Loop over layers
        for (unsigned int layer = 0; layer < numLayers; ++layer) {
			CSCMatrix W_d;
			unsigned int* w_colPtrs_d;
			unsigned int* w_rowIdxs_d;
			float* w_values_d;
			hipMalloc((void**)&W_d, sizeof(CSCMatrix));
			hipMalloc((void**)&w_colPtrs_d, (W[layer]->numCols + 1)* sizeof(unsigned int));
			hipMalloc((void**)&w_rowIdxs_d, W[layer]->numRows * sizeof(unsigned int));
			hipMalloc((void**)&w_values_d, W[layer]->numRows * sizeof(float));
			//copying W_d[layer]
			hipMemcpy(&W_d, W[layer], sizeof(CSCMatrix), hipMemcpyHostToDevice);
			hipMemcpy(w_colPtrs_d, W[layer]->colPtrs, W[layer]->numCols * sizeof(unsigned int), hipMemcpyHostToDevice);
			hipMemcpy(w_rowIdxs_d, W[layer]->rowIdxs, W[layer]->numRows * sizeof(unsigned int), hipMemcpyHostToDevice);
			hipMemcpy(w_values_d, W[layer]->values, W[layer]->numRows * sizeof(float), hipMemcpyHostToDevice);
			hipMemcpy(&(W_d.colPtrs), &w_colPtrs_d, sizeof(unsigned int*), hipMemcpyHostToDevice);
			hipMemcpy(&(W_d.rowIdxs), &w_rowIdxs_d, sizeof(unsigned int*), hipMemcpyHostToDevice);
			hipMemcpy(&(W_d.values), &w_values_d, sizeof(float*), hipMemcpyHostToDevice);
                // SpMSpM
                printf("Computing layer %u (SpMSpM)", layer);
                startTime(&timer);
                //unsigned int nnzIdx=0;

                //do kernel call instead
                //int outputSize = inBuffer_d->numRows * W_d[layer]->numCols;

                dim3 numThreadsPerBlock(threads, threads);
                dim3 numBlocks((W_d.numCols + numThreadsPerBlock.x - 1)/numThreadsPerBlock.x,(inBuffer_d.numRows + numThreadsPerBlock.y - 1)/numThreadsPerBlock.y);
                //int numBlocks = (outputSize + numThreadsPerBlock - 1)/numThreadsPerBlock ;
                spmspm <<<numBlocks, numThreadsPerBlock>>> (outBuffer_d, inBuffer_d, W_d, bias);
                printf("iiiiiii");
                //printf("size of outbuffer %d", outBuffer_d->nnz);
                hipDeviceSynchronize();
                stopTimeAndPrint(&timer, "");

                stopTimeAndPrint(&timer, "For Out Buffer");
				
				unsigned int* out_rowIdxs_h = (unsigned int*) malloc (outBuffer->capacity * sizeof(unsigned int));
				unsigned int* out_colIdxs_h = (unsigned int*) malloc (outBuffer->capacity * sizeof(unsigned int));;
				float* out_values_h = (float*) malloc (outBuffer->capacity * sizeof(unsigned int));;
				
                hipMemcpy(outBuffer, &outBuffer_d, sizeof(COOMatrix), hipMemcpyDeviceToHost);
		//struct fields as variables(?)
				hipMemcpy(out_rowIdxs_h, outBuffer_d->rowIdxs, outBuffer_d->capacity * sizeof(unsigned int), hipMemcpyDeviceToHost);
				hipMemcpy(out_colIdxs_h, outBuffer_d->colIdxs, outBuffer_d->capacity * sizeof(unsigned int), hipMemcpyDeviceToHost);
				hipMemcpy(out_values_h, outBuffer_d->values, outBuffer_d->capacity * sizeof(float), hipMemcpyDeviceToHost);
		//copy pointers back (??)
				hipMemcpy(&out_rowIdxs_h, &(outBuffer->rowIdxs), sizeof(unsigned int*), hipMemcpyDeviceToHost);
				hipMemcpy(&out_colIdxs_h, &(outBuffer->colIdxs), sizeof(unsigned int*), hipMemcpyDeviceToHost);
				hipMemcpy(&out_values_h, &(outBuffer->values), sizeof(float*), hipMemcpyDeviceToHost);
				printf("%f \n", outBuffer->values[0]);
				break;
		//stopTimeAndPrint(&timer, "For Sort");
        //inBuffer = createCSRfromCOO(sortCOO(outBuffer));
        //stopTimeAndPrint(&timer, "Out of sort");


                // outBuffer = createEmptyCOO(inBuffer->numRows, inBuffer->numCols, 2*inBuffer->capacity);
                // outBuffer_d.numRows = outBuffer->numRows;
                // outBuffer_d.numCols = outBuffer->numCols;
                // outBuffer_d.nnz = outBuffer->nnz;
                // outBuffer_d.capacity = outBuffer->capacity;
                // hipMemcpy(outBuffer_d.rowIdxs, outBuffer->rowIdxs, outBuffer_d.capacity * sizeof(unsigned int), hipMemcpyHostToDevice);
                // hipMemcpy(outBuffer_d.colIdxs, outBuffer->colIdxs, outBuffer_d.capacity         * sizeof(unsigned int), hipMemcpyHostToDevice);
                // hipMemcpy(outBuffer_d.values, outBuffer->values, outBuffer_d.capacity * sizeof(float), hipMemcpyHostToDevice);

        }

/////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
/////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
/////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
/////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////



        // Copy data from GPU
        startTime(&timer);

        // TODO

        hipMemcpy(inBuffer->rowPtrs, inBuffer_d.rowPtrs, inBuffer_d.numRows * sizeof(unsigned int), hipMemcpyDeviceToHost);
        hipMemcpy(inBuffer->colIdxs, inBuffer_d.colIdxs, inBuffer_d.numCols * sizeof(unsigned int), hipMemcpyDeviceToHost);
        hipMemcpy(inBuffer->values, inBuffer_d.values, inBuffer_d.numCols * sizeof(float), hipMemcpyDeviceToHost);


        hipDeviceSynchronize();
        stopTime(&timer);
        printElapsedTime(timer, "Copy from GPU time");

        //CPU
        // Find nonzero rows
        startTime(&timer);
        findNonzeroRows(result, inBuffer);
      stopTimeAndPrint(&timer, "Find nonzero rows");

        // Free GPU memory
 
      


}
